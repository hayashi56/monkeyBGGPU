#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "param.h"

__global__ void updateSynapse_MSN_D1 ( int nt, neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){
        n_MSN_D1 -> psp_GABA[ i ] = ( float ) tauGABA * n_MSN_D1 -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_MSN_D1 -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_MSN_D2 ( int nt, neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){
        n_MSN_D2 -> psp_GABA[ i ] = ( float ) tauGABA * n_MSN_D2 -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_MSN_D2 -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){
        n_FSI -> psp_GABA[ i ] = ( float ) tauGABA * n_FSI -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_FSI -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){
        n_STN -> psp_AMPA[ i ] = ( float ) tauAMPA * n_STN -> psp_AMPA[ i ] + PSP_amplitudes_AMPA * ( n_STN -> ts[ i ] + DELAY == nt );
        n_STN -> psp_NMDA[ i ] = ( float ) tauNMDA * n_STN -> psp_NMDA[ i ] + PSP_amplitudes_NMDA * ( n_STN -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){
        n_GPe -> psp_GABA[ i ] = ( float ) tauGABA * n_GPe -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_GPe -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_GPi ( int nt, neuron_t *n_GPi ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){
        n_GPi -> psp_GABA[ i ] = ( float ) tauGABA * n_GPi -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_GPi -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_SNc ( int nt, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){
        n_SNc -> psp_DOPA[ i ] = ( float ) tauDOPA * n_SNc -> psp_DOPA[ i ] + PSP_amplitudes_DOPA * ( n_SNc -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_PTN ( int nt, neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_PTN ){
        n_PTN -> psp_AMPA[ i ] = ( float ) tauAMPA * n_PTN -> psp_AMPA[ i ] + PSP_amplitudes_AMPA * ( n_PTN -> ts[ i ] + DELAY == nt );
        n_PTN -> psp_NMDA[ i ] = ( float ) tauNMDA * n_PTN -> psp_NMDA[ i ] + PSP_amplitudes_NMDA * ( n_PTN -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_PTI ( int nt, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){
        n_PTI -> psp_GABA[ i ] = ( float ) tauGABA * n_PTI -> psp_GABA[ i ] + PSP_amplitudes_GABA * ( n_PTI -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_PSN ( int nt, neuron_t *n_PSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_PSN ){
        n_PSN -> psp_AMPA[ i ] = ( float ) tauAMPA * n_PSN -> psp_AMPA[ i ] + PSP_amplitudes_AMPA * ( n_PSN -> ts[ i ] + DELAY == nt );
        n_PSN -> psp_NMDA[ i ] = ( float ) tauNMDA * n_PSN -> psp_NMDA[ i ] + PSP_amplitudes_NMDA * ( n_PSN -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_Th ( int nt, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_Th ){
        n_Th -> psp_AMPA[ i ] = ( float ) tauAMPA * n_Th -> psp_AMPA[ i ] + PSP_amplitudes_AMPA * ( n_Th -> ts[ i ] + DELAY == nt );
        n_Th -> psp_NMDA[ i ] = ( float ) tauNMDA * n_Th -> psp_NMDA[ i ] + PSP_amplitudes_NMDA * ( n_Th -> ts[ i ] + DELAY == nt );
    }
}
__global__ void updateSynapse_CMPf ( int nt, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CMPf ){
        n_CMPf -> psp_AMPA[ i ] = ( float ) tauAMPA * n_CMPf -> psp_AMPA[ i ] + PSP_amplitudes_AMPA * ( n_CMPf -> ts[ i ] + DELAY == nt );
        n_CMPf -> psp_NMDA[ i ] = ( float ) tauNMDA * n_CMPf -> psp_NMDA[ i ] + PSP_amplitudes_NMDA * ( n_CMPf -> ts[ i ] + DELAY == nt );
    }
}
void updateSynapse ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    updateSynapse_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( nt, n_MSN_D1 );
    hipDeviceSynchronize ( );
    updateSynapse_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( nt, n_MSN_D2 );
    hipDeviceSynchronize ( );
    updateSynapse_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    updateSynapse_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    updateSynapse_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    updateSynapse_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_GPi );
    hipDeviceSynchronize ( );
    updateSynapse_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( nt, n_SNc );
    hipDeviceSynchronize ( );
    updateSynapse_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN );
    hipDeviceSynchronize ( );
    updateSynapse_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( nt, n_PTI );
    hipDeviceSynchronize ( );
    updateSynapse_PSN <<< GRID_SIZE_PSN, BLOCK_SIZE >>> ( nt, n_PSN );
    hipDeviceSynchronize ( );
    updateSynapse_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( nt, n_Th );
    hipDeviceSynchronize ( );
    updateSynapse_CMPf <<< GRID_SIZE_CMPf, BLOCK_SIZE >>> ( nt, n_CMPf );
    hipDeviceSynchronize ( );
}

__global__ void InputSynapsePotential_MSN_D1 ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_MSN_D1 ){
        float r = 0;
        int t = 0;
        // x =MSN_D1;
        // y =CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){
            r += ( n_CMPf ->  psp_AMPA[ j ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_AMPA ) * W_CMPfMSND1_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_NMDA ) * W_CMPfMSND1_NMDA );
        }
        t++;
        // y =MSN_D1;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){
            r += n_MSN_D1 -> psp_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_GABA ) * W_MSND1MSND1;
        }
        t++;
        // y =MSN_D2;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){
            r += n_MSN_D2 -> psp_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_GABA ) * W_MSND2MSND1;
        }
        t++;
        // y =FSI;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += n_FSI -> psp_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseFSI ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_GABA ) * W_FSIMSND1;
        }
        t++;

        // y =STN;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += ( n_STN -> psp_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSTN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_AMPA ) * W_STNMSND1_AMPA ) + ( n_STN ->  psp_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSTN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_NMDA ) * W_STNMSND1_NMDA );
        }
        t++;

        // y =GPe;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseGPe ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_GABA ) * W_GPeMSND1;
        }
        t++;

        // y =SNc;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += n_SNc -> psp_DOPA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSNc ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_DOPA ) * W_SNcMSND1;
        }
        t++;

        // y =PTN;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePTN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_AMPA ) * W_PTNMSND1_AMPA ) + ( n_PTN ->  psp_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePTN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_NMDA ) * W_PTNMSND1_NMDA );
        }
        t++;

        // y =PSN;
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += ( n_PSN -> psp_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePSN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_AMPA ) * W_PSNMSND1_AMPA ) + ( n_PSN ->  psp_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePSN ] * ( - n_MSN_D1 -> v[ i ] + rev_potential_NMDA ) * W_PSNMSND1_NMDA );
        }
        n_MSN_D1 -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_MSN_D2 ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_MSN_D2 ){
        float r = 0;
        int t = 0;
        // x =MSN_D2;
        // y =CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){
            r += ( n_CMPf -> psp_AMPA[ j ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_AMPA ) * W_CMPfMSND2_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_NMDA ) * W_CMPfMSND2_NMDA );
        }
        n_MSN_D2 -> k [ i ] = r;
        t++;
        // y =MSN_D1;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){
            r += n_MSN_D1 -> psp_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_GABA ) * W_MSND1MSND2;
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 ] = r;
        t++;
        // y =MSN_D2;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){
            r += n_MSN_D2 -> psp_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_GABA ) * W_MSND2MSND2;
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 2 ] = r;
        t++;
        // y =FSI;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += n_FSI -> psp_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseFSI ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_GABA ) * W_FSIMSND2;
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 3 ] = r;
        t++;

        // y =STN;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += ( n_STN -> psp_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSTN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_AMPA ) * W_STNMSND2_AMPA ) + ( n_STN ->  psp_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSTN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_NMDA ) * W_STNMSND2_NMDA );
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 4 ] = r;
        t++;

        // y =GPe;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseGPe ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_GABA ) * W_GPeMSND2;
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 5 ] = r;
        t++;

        // y =SNc;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += n_SNc -> psp_DOPA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSNc ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_DOPA ) * W_SNcMSND2;
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 6 ] = r;
        t++;

        // y =PTN;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePTN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_AMPA ) * W_PTNMSND2_AMPA ) + ( n_PTN ->  psp_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePTN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_NMDA ) * W_PTNMSND2_NMDA );
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 7 ] = r;
        t++;

        // y =PSN;
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += ( n_PSN -> psp_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePSN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_AMPA ) * W_PSNMSND2_AMPA ) + ( n_PSN ->  psp_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePSN ] * ( - n_MSN_D2 -> v[ i ] + rev_potential_NMDA ) * W_PSNMSND2_NMDA );
        }
        n_MSN_D2 -> k [ i + N_MSN_D2 * 8 ] = r;
        n_MSN_D2 -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_FSI ( int nt, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){
    
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_FSI ) {
        float r = 0;
        int t = 0;
        // x =FSI;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( n_CMPf -> psp_AMPA[ j ] * ( - n_FSI -> v[ i ] + rev_potential_AMPA ) * W_CMPfFSI_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_FSI -> v[ i ] + rev_potential_NMDA ) * W_CMPfFSI_NMDA );
        }
        t++;
        // y =FSI;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += n_FSI -> psp_GABA[ n_FSI -> post[ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseFSI ] * ( - n_FSI -> v[ i ] + rev_potential_GABA ) * W_FSIFSI;
        }
        t++;
        // y =STN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( n_STN -> psp_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] * ( - n_FSI -> v[ i ] + rev_potential_AMPA ) * W_STNFSI_AMPA ) + ( n_STN ->  psp_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] * ( - n_FSI -> v[ i ] + rev_potential_NMDA ) * W_STNFSI_NMDA );
        }
        t++;
        // y =GPe;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_FSI -> post[ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseGPe ] * ( - n_FSI -> v[ i ] + rev_potential_GABA ) * W_GPeFSI;
        }
        t++;
        // y =PTN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] * ( - n_FSI -> v[ i ] + rev_potential_AMPA ) * W_PTNFSI_AMPA ) + ( n_PTN ->  psp_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] * ( - n_FSI -> v[ i ] + rev_potential_NMDA ) * W_PTNFSI_NMDA );
        }
        t++;
        // y =PSN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( n_PSN -> psp_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePSN ] * ( - n_FSI -> v[ i ] + rev_potential_AMPA ) * W_PSNFSI_AMPA ) + ( n_PSN ->  psp_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePSN ] * ( - n_FSI -> v[ i ] + rev_potential_NMDA ) * W_PSNFSI_NMDA );
        }
        n_FSI -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_STN ( int nt, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_PTN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_STN ){
        float r = 0;
        int t = 0;
        // x =STN;
        // y =CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( n_CMPf -> psp_AMPA[ j ] * ( - n_STN -> v[ i ] + rev_potential_AMPA ) * W_CMPfSTN_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_STN -> v[ i ] + rev_potential_NMDA ) * W_CMPfSTN_NMDA );
        }
        t++;
        // y =GPe;
        for ( int j = 0; j < ( n_STN -> num_pre[ i + ( t * N_STN ) ] - n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_STN -> post[ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapseGPe ] * ( - n_STN -> v[ i ] + rev_potential_GABA ) * W_GPeSTN;
        }
        t++;
        // y =PTN;
        for ( int j = 0; j < ( n_STN -> num_pre[ i + ( t * N_STN ) ] - n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] * ( - n_STN -> v[ i ] + rev_potential_AMPA ) * W_PTNSTN_AMPA ) + ( n_PTN ->  psp_NMDA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] * ( - n_STN -> v[ i ] + rev_potential_NMDA ) * W_PTNSTN_NMDA );
        }
        t++;
        n_STN -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_GPe ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_GPe ){
        float r = 0;
        int t = 0;
        // x =GPe;
        // y =CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( n_CMPf -> psp_AMPA[ j ] * ( - n_GPe -> v[ i ] + rev_potential_AMPA ) * W_CMPfGPe_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_GPe -> v[ i ] + rev_potential_NMDA ) * W_CMPfGPe_NMDA );
        }
        t++;
        // y =MSN_D1;
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){
            r += n_MSN_D1 -> psp_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_MSND1GPe;
        }
        t++;
        // y =MSN_D2;
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){
            r += n_MSN_D2 -> psp_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_MSND2GPe;
        }
        t++;
        // y =STN;
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r += ( n_STN -> psp_AMPA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] * ( - n_GPe -> v[ i ] + rev_potential_AMPA ) * W_STNGPe_AMPA ) + ( n_STN ->  psp_NMDA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] * ( - n_GPe -> v[ i ] + rev_potential_NMDA ) * W_STNGPe_NMDA );
        }
        t++;
        // y =GPe;
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseGPe ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_GPeGPe;
        }
        n_GPe -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_GPi ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_GPi ){
        float r = 0;
        int t = 0;
        // x =GPi;
        // y =CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( n_CMPf -> psp_AMPA[ j ] * ( - n_GPi -> v[ i ] + rev_potential_AMPA ) * W_CMPfGPi_AMPA ) + ( n_CMPf ->  psp_NMDA[ j ] * ( - n_GPi -> v[ i ] + rev_potential_NMDA ) * W_CMPfGPi_NMDA );
        }
        t++;
        // y =MSN_D1;
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){
            r += n_MSN_D1 -> psp_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_MSND1GPi;
        }
        t++;
        // y =MSN_D2;
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){
            r += n_MSN_D2 -> psp_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_MSND2GPi;
        }
        t++;
        // y =STN;
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r += ( n_STN -> psp_AMPA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] * ( - n_GPi -> v[ i ] + rev_potential_AMPA ) * W_STNGPi_AMPA ) + ( n_STN ->  psp_NMDA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] * ( - n_GPi -> v[ i ] + rev_potential_NMDA ) * W_STNGPi_NMDA );
        }
        t++;
        // y =GPe;
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r += n_GPe -> psp_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseGPe ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_GPeGPi;
        }
        n_GPi -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_SNc ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;
    int t = 0;

    if ( i < N_SNc ){
        float r = 0;
        // x =SNc;
        // y =MSN_D1;
        for ( int j = 0; j < ( n_SNc -> num_pre[ i + 1 ] - n_SNc -> num_pre[ i ] ); j++ ){ 
            r += n_MSN_D1 -> psp_GABA[ n_SNc -> post[ j + n_SNc -> num_pre[ i ] ] - SynapseMSN_D1 ] * ( - n_SNc -> v[ i ] + rev_potential_GABA ) * W_MSND1SNc;
        }
        t++;
        // y =MSN_D2;
        for ( int j = 0; j < ( n_SNc -> num_pre[ i + ( t * N_SNc ) + 1 ] - n_SNc -> num_pre[ i + ( t * N_SNc ) ] ); j++ ){
            r += n_MSN_D2 -> psp_GABA[ n_SNc -> post[ j + n_SNc -> num_pre[ i + ( t * N_SNc ) ] ] - SynapseMSN_D2 ] * ( - n_SNc -> v[ i ] + rev_potential_GABA ) * W_MSND2SNc;
        }
        n_SNc -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_PTN ( int nt, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;
    if ( i < N_PTN ){
        float r = 0;
        int t = 0;
        // x =PTN;
        // y =PTI;
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + 1 ] - n_PTN -> num_pre[ i ] ); j++ ){
            r += n_PTI -> psp_GABA[ n_PTN -> post[ j + n_PTN -> num_pre[ i ] ] - SynapsePTI ] * ( - n_PTN -> v[ i ] + rev_potential_GABA ) * W_PTIPTN;
        }
        t++;
        // y =PSN;
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + ( t * N_PTN ) + 1 ] - n_PTN -> num_pre[ i + ( t * N_PTN ) ] ); j++ ){ 
            r += ( n_PSN -> psp_AMPA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapsePSN ] * ( - n_PTN -> v[ i ] + rev_potential_AMPA ) * W_PSNPTN_AMPA ) + ( n_PSN ->  psp_NMDA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapsePSN ] * ( - n_PTN -> v[ i ] + rev_potential_NMDA ) * W_PSNPTN_NMDA );
        }
        t++;
        // y =Th;
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + ( t * N_PTN ) + 1 ] - n_PTN -> num_pre[ i + ( t * N_PTN ) ] ); j++ ){ 
            r += ( n_Th -> psp_AMPA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapseTh ] * ( - n_PTN -> v[ i ] + rev_potential_AMPA ) * W_ThPTN_AMPA ) + ( n_Th ->  psp_NMDA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapseTh ] * ( - n_PTN -> v[ i ] + rev_potential_NMDA ) * W_ThPTN_NMDA );
        }
        n_PTN -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_PTI ( int nt, neuron_t *n_PTN, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_PTI ){
        float r = 0;
        // x =PTI;
        // y =PTN;
        for ( int j = 0; j < ( n_PTI -> num_pre[ i + 1 ] - n_PTI -> num_pre[ i ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_PTI -> post [ j + n_PTI -> num_pre[ i ] ] - SynapsePTN ] * ( - n_PTI -> v[ i ] + rev_potential_AMPA ) * W_PTNPTI_AMPA ) + ( n_PTN ->  psp_NMDA[ n_PTI -> post [ j + n_PTI -> num_pre[ i ] ] - SynapsePTN ] * ( - n_PTI -> v[ i ] + rev_potential_NMDA ) * W_PTNPTI_NMDA );
        }
        n_PTI -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_Th ( int nt, neuron_t *n_GPi, neuron_t *n_PTN, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    // neuron_kind x, y;

    if ( i < N_Th ){
        float r = 0;
        int t = 0;
        // x =Th;
        // y =GPi;
        for ( int j = 0; j < ( n_Th -> num_pre[ i + 1 ] - n_Th -> num_pre[ i ] ); j++ ){
            r += n_GPi -> psp_GABA[ n_Th -> post[ j + n_Th -> num_pre[ i ] ] - SynapseGPi ] * ( - n_Th -> v[ i ] + rev_potential_GABA ) * W_GPiTh;
        }
        t++;
        // y =PTN;
        for ( int j = 0; j < ( n_Th -> num_pre[ i + ( t * N_Th ) + 1 ] - n_Th -> num_pre[ i + ( t * N_Th ) ] ); j++ ){ 
            r += ( n_PTN -> psp_AMPA[ n_Th -> post [ j + n_Th -> num_pre[ i + ( t * N_Th ) - 1 ] ] - SynapsePTN ] * ( - n_Th -> v[ i ] + rev_potential_AMPA ) * W_PTNTh_AMPA ) + ( n_PTN ->  psp_NMDA[ n_Th -> post [ j + n_Th -> num_pre[ i + ( t * N_Th ) - 1 ] ] - SynapsePTN ] * ( - n_Th -> v[ i ] + rev_potential_NMDA ) * W_PTNTh_NMDA );
        }
        n_Th -> ig[ i ] = r;
    }
}

void InputSynapsePotential ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    InputSynapsePotential_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_SNc, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_SNc, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI, n_STN, n_GPe, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN, n_GPe, n_PTN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_MSN_D1, n_MSN_D2, n_STN, n_GPe, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_MSN_D1, n_MSN_D2, n_STN, n_GPe, n_GPi, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( nt, n_MSN_D1, n_MSN_D2, n_SNc );
    hipDeviceSynchronize ( );
    InputSynapsePotential_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN, n_PTI, n_PSN, n_Th );
    hipDeviceSynchronize ( );
    InputSynapsePotential_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( nt, n_PTN, n_PTI );
    hipDeviceSynchronize ( );
    InputSynapsePotential_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( nt, n_GPi, n_PTN, n_Th );
    hipDeviceSynchronize ( );
}