#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include "param.h"
#include "structure.cu"
#include "initalize.cu"
#include "synapse_function.cu"
#include "update.cu"
#include "output.cu"
#include "function.cu"

int main( void ){

  neuron_t *msn, *fsi, *stn, *gpe, *gpi, *csn, *ptn, *cmpf;

  //初期化・メモリの確保・ファイルを開く
  hipMallocManaged ( &msn, sizeof ( neuron_t ) );
  hipMallocManaged ( &fsi, sizeof ( neuron_t ) );
  hipMallocManaged ( &stn, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpe, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpi, sizeof ( neuron_t ) );
  hipMallocManaged ( &csn, sizeof ( neuron_t ) );
  hipMallocManaged ( &ptn, sizeof ( neuron_t ) );
  hipMallocManaged ( &cmpf, sizeof ( neuron_t ) );
  
  initalizeNeuron ( msn, fsi, stn, gpe, gpi, csn, ptn, cmpf );
  initsynapse ( msn, fsi, stn, gpe, gpi );

  for ( int w = 0; w < N_i; w++ ){
    for ( int z = 0; z < N_i; z++ ){
      loop ( msn, fsi, stn, gpe, gpi, csn, ptn, cmpf, w, z );
    }
  }

  //メモリの解放・ファイルを閉じる
  fileclose ( gpi, csn, ptn );
  finalize ( msn, fsi, stn, gpe, gpi, csn, ptn, cmpf );

  hipFree ( msn );
  hipFree ( fsi );
  hipFree ( stn );
  hipFree ( gpe );
  hipFree ( gpi );
  hipFree ( csn );
  hipFree ( ptn );
  hipFree ( cmpf );

  return 0;
}