#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>

extern "C" { void timer_start( void ); }
extern "C" { double timer_elapsed( void ); }

//不応期
// MSN
__global__ void t_refr_MSN ( neuron_t *n_MSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN ){

        n_MSN -> refr [ i ] = ( n_MSN -> s [ i ] ) * ( T_REFR ) + ( !( n_MSN -> s [ i ] ) ) * ( ( n_MSN -> refr [ i ] ) - 1 ); // set counter
        n_MSN -> v [ i ] = ( ( n_MSN -> refr [ i ] ) > 0 ) ? ( V_RESET ) : ( ( n_MSN -> v [ i ] ) );
    }
}
// FSI
__global__ void t_refr_FSI ( neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> refr [ i ] = ( n_FSI -> s [ i ] ) * ( T_REFR ) + ( !( n_FSI -> s [ i ] ) ) * ( ( n_FSI -> refr [ i ] ) - 1 ); // set counter
        n_FSI -> v [ i ] = ( ( n_FSI -> refr [ i ] ) > 0 ) ? ( V_RESET ) : ( n_FSI -> v [ i ] );
    }
}
// STN
__global__ void t_refr_STN ( neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> refr [ i ] = ( n_STN -> s [ i ] ) * ( T_REFR ) + ( !( n_STN -> s [ i ] ) ) * ( ( n_STN -> refr [ i ] ) - 1 ); // set counter
        n_STN -> v [ i ] = ( ( n_STN -> refr [ i ] ) > 0 ) ? ( V_RESET ) : ( n_STN -> v [ i ] );
    }
}
// GPe
__global__ void t_refr_GPe ( neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> refr [ i ] = ( n_GPe -> s [ i ] ) * ( T_REFR ) + ( !( n_GPe -> s [ i ] ) ) * ( ( n_GPe -> refr [ i ] ) - 1 ); // set counter
        n_GPe -> v [ i ] = ( ( n_GPe -> refr [ i ] ) > 0) ? ( V_RESET ) : ( n_GPe -> v [ i ] );
    }
}
// GPi
__global__ void t_refr_GPi ( neuron_t *n_GPi ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> refr [ i ] = ( n_GPi -> s [ i ] ) * ( T_REFR ) + ( !( n_GPi -> s [ i ] ) ) * ( ( n_GPi -> refr [ i ] ) - 1 ); // set counter
        n_GPi -> v [ i ] = ( ( n_GPi -> refr [ i ] ) > 0 ) ? ( V_RESET ) : ( n_GPi -> v [ i ] );
    }
}
// CSN
__global__ void t_refr_CSN ( neuron_t *n_CSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CSN ){

        n_CSN -> refr [ i ] = ( n_CSN -> s [ i ] ) * ( T_REFR ) + ( !( n_CSN -> s [ i ] ) ) * ( ( n_CSN -> refr [ i ] ) - 1 ); // set counter
    }
}
// PTN
__global__ void t_refr_PTN ( neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTN ){

        n_PTN -> refr [ i ] = ( n_PTN -> s [ i ] ) * ( T_REFR ) + ( !( n_PTN -> s [ i ] ) ) * ( ( n_PTN -> refr [ i ] ) - 1 ); // set counter
    }
}
// CMPf
__global__ void t_refr_CMPf ( neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CMPf ){

        n_CMPf -> refr [ i ] = ( n_CMPf -> s [ i ] ) * ( T_REFR ) + ( ! ( n_CMPf -> s [ i ] ) ) * ( ( n_CMPf -> refr [ i ] ) - 1 ); // set counter
    }
}
void t_refr ( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    t_refr_MSN <<< GRID_SIZE_MSN, BLOCK_SIZE >>> ( n_MSN );
    hipDeviceSynchronize ( );
    t_refr_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( n_FSI );
    hipDeviceSynchronize ( );
    t_refr_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( n_STN );
    hipDeviceSynchronize ( );
    t_refr_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( n_GPe );
    hipDeviceSynchronize ( );
    t_refr_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( n_GPi );
    hipDeviceSynchronize ( );
    t_refr_CSN <<< GRID_SIZE_CSN, BLOCK_SIZE >>> ( n_CSN );
    hipDeviceSynchronize ( );
    t_refr_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( n_PTN );
    hipDeviceSynchronize ( );
    t_refr_CMPf <<< GRID_SIZE_CMPf, BLOCK_SIZE >>> ( n_CMPf );
    hipDeviceSynchronize ( );
}

//ループ
void loop( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf, int w, int z ){

    timer_start();

    for ( int nt = 0; nt < FreeRun; nt++ ){
        updateSynapse ( nt, n_MSN, n_FSI, n_STN, n_GPe, n_CSN, n_PTN, n_CMPf );
        InputSynapsePotential ( nt, n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
        updatePotential( nt, n_MSN, n_FSI, n_STN, n_GPe, n_GPi );
        inputneuron ( nt, n_CSN, n_PTN, n_CMPf );
        t_refr ( n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
    }

    for ( int nt = 0; nt < NT; nt++ ){

        updateSynapse ( nt, n_MSN, n_FSI, n_STN, n_GPe, n_CSN, n_PTN, n_CMPf );
        InputSynapsePotential ( nt, n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
        updatePotential( nt, n_MSN, n_FSI, n_STN, n_GPe, n_GPi );
        change_inputneuron ( nt, n_CSN, n_PTN, n_CMPf, w, z );
        t_refr ( n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
        outputSpike ( n_GPi, n_CSN, n_PTN );
    }

    double elapsedTime = timer_elapsed ();

    printf ( "Elapsed time = %f sec.\n", elapsedTime );

    outputFiringRate ( n_GPi, n_CSN, n_PTN );

    for ( int i = 0; i < N_MSN; i++ ){
        n_MSN -> ig[ i ] = 0;
        n_MSN -> alpha_GABA[ i ] = 0;
        n_MSN -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_MSN -> rng );
        n_MSN -> s[ i ] = false;
        n_MSN -> ts[ i ] = 1000;
        n_MSN -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_FSI; i++ ){
        n_FSI -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_FSI -> rng );
        n_FSI -> s[ i ] = false;
        n_FSI -> alpha_GABA[ i ] = 0;
        n_FSI -> ts[ i ] = 1000;
        n_FSI -> ig[ i ] = 0;
        n_FSI -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_GPi; i++ ){
        n_GPi -> ig[ i ] = 0;
        n_GPi -> counter[ i ] = 0;
        n_GPi -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_GPi -> rng );
        n_GPi -> s[ i ] = false;
        n_GPi -> ts[ i ] = 1000;
        n_GPi -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_STN; i++ ){
        n_STN -> ig[ i ] = 0;
        n_STN -> alpha_AMPA[ i ] = 0;
        n_STN -> alpha_NMDA[ i ] = 0;
        n_STN -> v[ i ] = V_INIT + 50. * sfmt_genrand_real2 ( &n_STN -> rng );
        n_STN -> s[ i ] = false;
        n_STN -> ts[ i ] = 1000;
        n_STN -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_GPe; i++ ){
        n_GPe -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_GPe -> rng );
        n_GPe -> s[ i ] = false;
        n_GPe -> alpha_GABA[ i ] = 0;
        n_GPe -> ts[ i ] = 1000;
        n_GPe -> ig[ i ] = 0;
        n_GPe -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_CSN; i++ ){
        n_CSN -> alpha_AMPA[ i ] = 0;
        n_CSN -> alpha_NMDA[ i ] = 0;
        n_CSN -> counter[ i ] = 0;
        n_CSN -> s[ i ] = false;
        n_CSN -> ts[ i ] = 1000;
        n_CSN -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_PTN; i++ ){
        n_PTN -> alpha_AMPA[ i ] = 0;
        n_PTN -> alpha_NMDA[ i ] = 0;
        n_PTN -> counter[ i ] = 0;
        n_PTN -> s[ i ] = false;
        n_PTN -> ts[ i ] = 1000;
        n_PTN -> refr[ i ] = 0;
    }

    for ( int i = 0; i < N_CMPf; i++ ){
        n_CMPf -> alpha_AMPA[ i ] = 0;
        n_CMPf -> alpha_NMDA[ i ] = 0;
        n_CMPf -> s[ i ] = false;
        n_CMPf -> ts[ i ] = 1000;
        n_CMPf -> refr[ i ] = 0;
    }

    initselection ( n_CSN, n_PTN );
}