#include <stdio.h>
#include <stdlib.h>
#include <SFMT.h>
#include <random>
#include <algorithm>
#include <vector>
#include "param.h"

void fileopen ( /*neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe,*/ neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN/*, neuron_t *n_CMPf*/ ){

    // n_MSN -> file = fopen ( "MSNspike.dat", "w" );
    // n_MSN -> file1 = fopen ( "MSNfiringrate.dat", "w" );
    // n_MSN -> file2 = fopen ( "MSN.dat", "w" );

    // n_FSI -> file = fopen ( "FSIspike.dat", "w" );
    // n_FSI -> file1 = fopen ( "FSIfiringrate.dat", "w" );
    // n_FSI -> file2 = fopen ( "FSI.dat", "w" );

    // n_STN -> file = fopen ( "STNspike.dat", "w" );
    // n_STN -> file1 = fopen ( "STNfiringrate.dat", "w" );
    // n_STN -> file2 = fopen ( "STN.dat", "w" );

    // n_GPe -> file = fopen ( "GPespike.dat", "w" );
    // n_GPe -> file1 = fopen ( "GPefiringrate.dat", "w" );
    // n_GPe -> file2 = fopen ( "GPe.dat", "w" );

    // n_GPi -> file = fopen ( "GPi.dat", "w" );
    n_GPi -> file1 = fopen ( "GPifiringrate_chanel1.dat", "w" );
    n_GPi -> file2 = fopen ( "GPifiringrate_chanel2.dat", "w" );

    //n_CSN -> file = fopen ( "CSNspike.dat", "w" );
    n_CSN -> file1 = fopen ( "CSNfiringrate_chanel1.dat", "w" );
    n_CSN -> file2 = fopen ( "CSNfiringrate_chanel2.dat", "w" );

    //n_PTN -> file = fopen ( "PTNspike.dat", "w" );
    n_PTN -> file1 = fopen ( "PTNfiringrate_chanel1.dat", "w" );
    n_PTN -> file2 = fopen ( "PTNfiringrate_chanel2.dat", "w" );

    // n_CMPf -> file = fopen ( "CMPfspike.dat", "w" );
    // n_CMPf -> file1 = fopen ( "CMPffiringrate.dat", "w" );
    // n_CMPf -> file2 = fopen ( "CMPf.dat", "w" );
}

void AllocatingNeuron ( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    hipMallocManaged ( &n_MSN -> v, sizeof ( float ) * N_MSN );
    hipMallocManaged ( &n_MSN -> ig, sizeof ( float ) * N_MSN );
    hipMallocManaged ( &n_MSN -> alpha_GABA, sizeof ( float ) * N_MSN );
    hipMallocManaged ( &n_MSN -> refr, sizeof ( int ) * N_MSN );
    hipMallocManaged ( &n_MSN -> ts, sizeof ( int ) * N_MSN );
    hipMallocManaged ( &n_MSN -> s, sizeof ( bool ) * N_MSN );
    // hipMallocManaged ( &n_MSN -> counter, sizeof ( int ) * N_MSN );
    hipMallocManaged ( &n_MSN -> num_pre, sizeof ( long ) * N_MSN * 7 );
    
    hipMallocManaged ( &n_FSI -> v, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> ig, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> alpha_GABA, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> s, sizeof ( bool ) * N_FSI );
    hipMallocManaged ( &n_FSI -> ts, sizeof ( int ) * N_FSI );
    hipMallocManaged ( &n_FSI -> refr, sizeof ( int ) * N_FSI );
    //hipMallocManaged ( &n_FSI -> counter, sizeof ( int ) * N_FSI );
    hipMallocManaged ( &n_FSI -> num_pre, sizeof ( long ) * N_FSI * 6 );
    
    hipMallocManaged ( &n_STN -> v, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> ig, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> alpha_AMPA, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> alpha_NMDA, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> s, sizeof ( bool ) * N_STN );
    hipMallocManaged ( &n_STN -> ts, sizeof ( int ) * N_STN );
    hipMallocManaged ( &n_STN -> refr, sizeof ( int ) * N_STN );
    //hipMallocManaged ( &n_STN -> counter, sizeof ( int ) * N_STN );
    hipMallocManaged ( &n_STN -> num_pre, sizeof ( long ) * N_STN * 3 );
    
    hipMallocManaged ( &n_GPe -> v, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> ig, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> alpha_GABA, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> s, sizeof ( bool ) * N_GPe );
    hipMallocManaged ( &n_GPe -> ts, sizeof ( int ) * N_GPe );
    hipMallocManaged ( &n_GPe -> refr, sizeof ( int ) * N_GPe );
    //hipMallocManaged ( &n_GPe -> counter, sizeof ( int ) * N_GPe );
    hipMallocManaged ( &n_GPe -> num_pre, sizeof ( long ) * N_GPe * 4 );

    hipMallocManaged ( &n_GPi -> v, sizeof ( float ) * N_GPi );
    hipMallocManaged ( &n_GPi -> ig, sizeof ( float ) * N_GPi );
    hipMallocManaged ( &n_GPi -> s, sizeof ( bool ) * N_GPi );
    hipMallocManaged ( &n_GPi -> ts, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> refr, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> counter, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> num_pre, sizeof ( long ) * N_GPi * 4 );

    hipMallocManaged ( &n_CSN -> alpha_AMPA, sizeof ( float ) * N_CSN );
    hipMallocManaged ( &n_CSN -> alpha_NMDA, sizeof ( float ) * N_CSN );
    hipMallocManaged ( &n_CSN -> s, sizeof ( bool ) * N_CSN );
    hipMallocManaged ( &n_CSN -> select, sizeof ( bool ) * N_CSN );
    hipMallocManaged ( &n_CSN -> ts, sizeof ( int ) * N_CSN );
    hipMallocManaged ( &n_CSN -> refr, sizeof ( int ) * N_CSN );
    hipMallocManaged ( &n_CSN -> counter, sizeof ( int ) * N_CSN );

    hipMallocManaged ( &n_PTN -> alpha_AMPA, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> alpha_NMDA, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> s, sizeof ( bool ) * N_PTN );
    hipMallocManaged ( &n_PTN -> select, sizeof ( bool ) * N_PTN );
    hipMallocManaged ( &n_PTN -> ts, sizeof ( int ) * N_PTN );
    hipMallocManaged ( &n_PTN -> refr, sizeof ( int ) * N_PTN );
    hipMallocManaged ( &n_PTN -> counter, sizeof ( int ) * N_PTN );

    hipMallocManaged ( &n_CMPf -> alpha_AMPA, sizeof ( float ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> alpha_NMDA, sizeof ( float ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> s, sizeof ( bool ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> ts, sizeof ( int ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> refr, sizeof ( int ) * N_CMPf );
    //hipMallocManaged ( &n_CMPf -> counter, sizeof ( int ) * N_CMPf );
}

void initNeuron ( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    for ( int i = 0; i < N_MSN; i++ ){

        n_MSN -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_MSN -> rng );
        n_MSN -> ig[ i ] = 0;
        n_MSN -> alpha_GABA[ i ] = 0;
        n_MSN -> refr[ i ] = 0;
        n_MSN -> ts[ i ] = 1000;
        n_MSN -> s[ i ] = false;
        //n_MSN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_FSI; i++ ){

        n_FSI -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_FSI -> rng );
        n_FSI -> ig[ i ] = 0;
        n_FSI -> alpha_GABA[ i ] = 0;
        n_FSI -> refr[ i ] = 0;
        n_FSI -> ts[ i ] = 1000;
        n_FSI -> s[ i ] = false;
        //n_FSI -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_STN; i++ ){

        n_STN -> v[ i ] = V_INIT + 50. * sfmt_genrand_real2 ( &n_STN -> rng );
        n_STN -> ig[ i ] = 0;
        n_STN -> alpha_AMPA[ i ] = 0;
        n_STN -> alpha_NMDA[ i ] = 0;
        n_STN -> refr[ i ] = 0;
        n_STN -> ts[ i ] = 1000;
        n_STN -> s[ i ] = false;
        //n_STN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_GPe; i++ ){

        n_GPe -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_GPe -> rng );
        n_GPe -> ig[ i ] = 0;
        n_GPe -> alpha_GABA[ i ] = 0;
        n_GPe -> refr[ i ] = 0;
        n_GPe -> ts[ i ] = 1000;
        n_GPe -> s[ i ] = false;
        //n_GPe -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_GPi; i++ ){

        n_GPi -> v[ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &n_GPi -> rng );
        n_GPi -> ig[ i ] = 0;
        n_GPi -> refr[ i ] = 0;
        n_GPi -> ts[ i ] = 1000;
        n_GPi -> s[ i ] = false;
        n_GPi -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_CSN; i++ ){

        n_CSN -> alpha_AMPA[ i ] = 0;
        n_CSN -> alpha_NMDA[ i ] = 0;
        n_CSN -> refr[ i ] = 0;
        n_CSN -> ts[ i ] = 1000;
        n_CSN -> s[ i ] = false;
        n_CSN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_PTN; i++ ){

        n_PTN -> alpha_AMPA[ i ] = 0;
        n_PTN -> alpha_NMDA[ i ] = 0;
        n_PTN -> refr[ i ] = 0;
        n_PTN -> ts[ i ] = 1000;
        n_PTN -> s[ i ] = false;
        n_PTN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_CMPf; i++ ){

        n_CMPf -> alpha_AMPA[ i ] = 0;
        n_CMPf -> alpha_NMDA[ i ] = 0;
        n_CMPf -> refr[ i ] = 0;
        n_CMPf -> ts[ i ] = 1000;
        n_CMPf -> s[ i ] = false;
        //n_CMPf -> counter[ i ] = 0;
    }
}

void initselection( neuron_t *n_CSN, neuron_t *n_PTN ){
    
    std::vector<int> v;
    for ( int i = 0; i != N_CSN; ++i ){
        v.push_back( i );
    }

    std::random_device get_rand_dev;
    std::mt19937 get_rand_mt ( get_rand_dev () );
    std::shuffle ( v.begin (), v.end (), get_rand_mt );

    for ( int i = 0; i < N_select; i++ ){
        n_CSN -> select [ v [ i ] ] = true;
        n_PTN -> select [ v [ i ] ] = true;
    }
}

void initalizeNeuron( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){
 
    fileopen ( /*n_MSN, n_FSI, n_STN, n_GPe,*/ n_GPi, n_CSN, n_PTN/*, n_CMPf*/ );
    AllocatingNeuron ( n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );

    // PRNG
    sfmt_init_gen_rand ( &( n_MSN -> rng ), 23 );
    sfmt_init_gen_rand ( &( n_FSI -> rng ), 22 );
    sfmt_init_gen_rand ( &( n_STN -> rng ), 21 );
    sfmt_init_gen_rand ( &( n_GPe -> rng ), 20 );
    sfmt_init_gen_rand ( &( n_GPi -> rng ), 19 );
    sfmt_init_gen_rand ( &( n_CSN -> rng ), 23 );
    sfmt_init_gen_rand ( &( n_PTN -> rng ), 23 );
    sfmt_init_gen_rand ( &( n_CMPf -> rng ), 23 );

    initNeuron ( n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
    initselection ( n_CSN, n_PTN );
}

void initsynapse( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi ){

    int k = 0;
    long num_of_synapse = 0;
    long post_synapse = 0;
    long count = 0;
    long count_pre = 0;

    srand ( 1 );
    // MSN→MSN(拡散型)
    num_of_synapse += N_MSN * N_MSN;
    post_synapse += N_MSN * N_MSN;

    // FSI→MSN(拡散型)
    num_of_synapse += N_FSI * N_MSN;
    post_synapse += N_FSI * N_MSN;

    // STN→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNMSN );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // GPe→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand ( ) / RAND_MAX ) < P_GPeMSN );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // CSN→MSN(集中型)
    num_of_synapse += per_chanel_N_CSN * per_chanel_N_MSN * N_chanel;
    post_synapse += per_chanel_N_CSN * per_chanel_N_MSN * N_chanel;

    // PTN→MSN(集中型)
    num_of_synapse += per_chanel_N_PTN * per_chanel_N_MSN * N_chanel;
    post_synapse += per_chanel_N_PTN * per_chanel_N_MSN * N_chanel;

    // CMPf→MSN(拡散型)
    num_of_synapse += N_CMPf * N_MSN;
    post_synapse += N_CMPf * N_MSN;

    hipMallocManaged ( &n_MSN -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // FSI→FSI(拡散型)
    num_of_synapse += N_FSI * N_FSI;
    post_synapse += N_FSI * N_FSI;

    // STN→FSI(拡散型)   
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNFSI );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // GPe→FSI(拡散型)    
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeFSI );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // CSN→FSI(集中型)
    num_of_synapse += per_chanel_N_CSN * per_chanel_N_FSI * N_chanel;
    post_synapse += per_chanel_N_CSN * per_chanel_N_FSI * N_chanel;

    // PTN→FSI(集中型)
    num_of_synapse += per_chanel_N_PTN * per_chanel_N_FSI * N_chanel;
    post_synapse += per_chanel_N_PTN * per_chanel_N_FSI * N_chanel;

    // CMPf→FSI(拡散型)
    num_of_synapse += N_CMPf * N_FSI;
    post_synapse += N_CMPf * N_FSI;

    hipMallocManaged ( &n_FSI -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // GPe→STN(集中型)
    num_of_synapse += per_chanel_N_GPe * per_chanel_N_STN * N_chanel;
    post_synapse += per_chanel_N_GPe * per_chanel_N_STN * N_chanel;

    // PTN→STN(集中型)
    num_of_synapse += per_chanel_N_PTN * per_chanel_N_STN * N_chanel;
    post_synapse += per_chanel_N_PTN * per_chanel_N_STN * N_chanel;

    // CMPf→STN(拡散型)
    num_of_synapse += N_CMPf * N_STN;
    post_synapse += N_CMPf * N_STN;

    hipMallocManaged ( &n_STN -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // MSN→GPe(集中型)
    num_of_synapse += per_chanel_N_MSN * per_chanel_N_GPe * N_chanel;
    post_synapse += per_chanel_N_MSN * per_chanel_N_GPe * N_chanel;

    // STN→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNGPe );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // GPe→GPe(拡散型)  
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPe );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // CMPf→GPe(拡散型)
    num_of_synapse += N_CMPf * N_GPe;
    post_synapse += N_CMPf * N_GPe;

    hipMallocManaged ( &n_GPe -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→GPi(拡散型)
    num_of_synapse += N_CMPf * N_GPi;
    post_synapse += N_CMPf * N_GPi;

    // STN→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNGPi );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // GPe→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPi );
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    // MSN→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSNGPi );
            }
        }
    }
    num_of_synapse += count;
    post_synapse += count;
    count = 0;

    hipMallocManaged ( &n_GPi -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;    


    printf("%ld\n",num_of_synapse);

    num_of_synapse = 0;

    srand ( 1 );

    // MSN→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){        
        for ( long j = 0; j < N_MSN; j++ ){
            n_MSN -> post[ count ] = j;
            count++;
            count_pre++;
        }
        n_MSN -> num_pre[ i + ( N_MSN * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // FSI→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){    
        for ( long j = 0; j < N_FSI; j++ ){
            n_MSN -> post[ count + num_of_synapse ] = j + SynapseFSI;
            count++;
            count_pre++;
        }
        n_MSN -> num_pre[ i + ( N_MSN * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // STN→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){    
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNMSN ){
                n_MSN -> post[ count + num_of_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_MSN -> num_pre[ i + ( N_MSN * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // GPe→MSN(拡散型)
    for ( long i = 0; i < N_MSN; i++ ){    
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeMSN ){
                n_MSN -> post[ count + num_of_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_MSN -> num_pre[ i + ( N_MSN * k ) ] = count_pre;  
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CSN→MSN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN; i++ ){
            for ( long j = 0; j < per_chanel_N_CSN; j++ ){
                n_MSN -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_CSN ) + SynapseCSN;
                count++;
                count_pre++;
            }
            n_MSN -> num_pre[ i + ( t * per_chanel_N_MSN ) + ( N_MSN * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // PTN→MSN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_MSN -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_MSN -> num_pre[ i + ( t * per_chanel_N_MSN ) + ( N_MSN * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CMPf→MSN(拡散型)    
    for ( long i = 0; i < N_MSN; i++ ){        
        for ( long j = 0; j < N_CMPf; j++ ){
            n_MSN -> post[ count + num_of_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_MSN -> num_pre[ i + ( N_MSN * k ) ] = count_pre;
    }
    num_of_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // FSI→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            n_FSI -> post[ count + num_of_synapse ] = j + SynapseFSI;
            count++;
            count_pre++;
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // STN→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){            
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNFSI ){
                n_FSI -> post[ count + num_of_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // GPe→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeFSI ){
                n_FSI -> post[ count + num_of_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CSN→FSI(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_FSI; i++ ){
            for ( long j = 0; j < per_chanel_N_CSN; j++ ){
                n_FSI -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_CSN ) + SynapseCSN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( t * per_chanel_N_FSI ) + ( N_FSI * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // PTN→FSI(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_FSI; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_FSI -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( t * per_chanel_N_FSI ) + ( N_FSI * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CMPf→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_FSI -> post[ count + num_of_synapse ] = j  + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    num_of_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // GPe→STN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_STN; i++ ){
            for ( long j = 0; j < per_chanel_N_GPe; j++ ){
                n_STN -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_GPe ) + SynapseGPe;
                count++;
                count_pre++;
            }
            n_STN -> num_pre[ i + ( t * per_chanel_N_STN ) + ( N_STN * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // PTN→STN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_STN; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_STN -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_STN -> num_pre[ i + ( t * per_chanel_N_STN ) + ( N_STN * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CMPf→STN(拡散型)
    for ( long i = 0; i < N_STN; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_STN -> post[ count + num_of_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_STN -> num_pre [ i + ( N_STN * k ) ] = count_pre;
    }
    num_of_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // MSN→GPe(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPe; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN; j++ ){
                n_GPe -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_MSN );
                count++;
                count_pre++;
            }
            n_GPe -> num_pre[ i + ( t * per_chanel_N_GPe ) + ( N_GPe * k ) ] = count_pre;
        }
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // STN→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){   
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNGPe ){
                n_GPe -> post[ count + num_of_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // GPe→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){     
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPe ){
                n_GPe -> post[ count + num_of_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // CMPf→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_GPe -> post[ count + num_of_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    num_of_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_GPi -> post[ count + num_of_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre;
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // STN→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNGPi ){
                n_GPi -> post[ count + num_of_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre; 
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // GPe→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPi ){
                n_GPi -> post[ count + num_of_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre; 
    }
    num_of_synapse += count;
    count = 0;
    k++;

    // MSN→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_MSNGPi ){
                    n_GPi -> post[ count + num_of_synapse ] = j + ( t * per_chanel_N_MSN );
                    count++;
                    count_pre++;
                }
            }
            n_GPi -> num_pre[ i + ( t * per_chanel_N_GPi ) + ( N_GPi * k ) ] = count_pre;
        }
    }
    num_of_synapse = 0;
    count = 0;
}