#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>

void outputFiringRate ( /*neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe,*/ neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN/*, neuron_t *n_CMPf */){

  float mean = 0;
  /*
  //MSN
  for ( int i = 0; i < N_MSN; i++ ){
    mean += n_MSN -> counter[ i ]; 
  }
  mean = mean / N_MSN;
  fprintf ( n_MSN -> file1, "%f\n", mean );

  //FSI
  mean = 0;
  for ( int i = 0; i < N_FSI; i++ ){
    mean += n_FSI -> counter[ i ];
  }
  mean = mean / N_FSI;
  fprintf ( n_FSI -> file1, "%f\n", mean );

  //STN
  mean = 0;
  for ( int i = 0; i < N_STN; i++ ){
    mean += n_STN -> counter[ i ];
  }
  mean = mean / N_STN;
  fprintf ( n_STN -> file1, "%f\n", mean );

  //GPe
  mean = 0;
  for ( int i = 0; i < N_GPe; i++ ){
    mean += n_GPe -> counter[ i ];
  }
  mean = mean / N_GPe;
  fprintf ( n_GPe -> file1, "%f\n", mean );

  */
  //GPi
  // mean = 0;
  for ( int i = 0; i < per_chanel_N_GPi; i++ ){
    mean += n_GPi -> counter[ i ];
  }
  mean = mean / per_chanel_N_GPi;
  fprintf ( n_GPi -> file1, "%f\n", mean );
  mean = 0;
  for ( int i = 0; i < per_chanel_N_GPi; i++ ){
    mean += n_GPi -> counter[ i + per_chanel_N_GPi ];
  }
  mean = mean / per_chanel_N_GPi;
  fprintf ( n_GPi -> file2, "%f\n", mean );

  //CSN
  mean = 0;
  for ( int i = 0; i < per_chanel_N_CSN; i++ ){
    mean += n_CSN -> counter[ i ];
  }
  mean = mean / per_chanel_N_CSN;
  fprintf ( n_CSN -> file1, "%f\n", mean );
  mean = 0;
  for ( int i = 0; i < per_chanel_N_CSN; i++ ){
    mean += n_CSN -> counter[ i + per_chanel_N_CSN ];
  }
  mean = mean / per_chanel_N_CSN;
  fprintf ( n_CSN -> file2, "%f\n", mean );

  //PTN
  mean = 0;
  for ( int i = 0; i < per_chanel_N_PTN; i++ ){
    mean += n_PTN -> counter[ i ];
  }
  mean = mean / per_chanel_N_PTN;
  fprintf ( n_PTN -> file1, "%f\n", mean );
  mean = 0;
  for ( int i = 0; i < per_chanel_N_PTN; i++ ){
    mean += n_PTN -> counter[ i + per_chanel_N_PTN ];
  }
  mean = mean / per_chanel_N_PTN;
  fprintf ( n_PTN -> file2, "%f\n", mean );

  /*
  //CMPf
  mean = 0;
  for ( int i = 0; i < N_CMPf; i++ ){
    mean += n_CMPf -> counter[ i ];
  }
  mean = mean / N_CMPf;
  fprintf ( n_CMPf -> file1, "%f\n", mean );
  */
}

void outputSpike ( /*int nt, neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe,*/ neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN/*, neuron_t *n_CMPf*/ ){

  /*
  //MSN
  for ( int i = 0; i < N_MSN; i++ ){

    if ( n_MSN -> s[ i ] ){

      fprintf ( n_MSN -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_MSN -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  //FSI
  for ( int i = 0; i < N_FSI; i++ ){

    if ( n_FSI -> s[ i ] ){

      fprintf ( n_FSI -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_FSI -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  //STN
  for ( int i = 0; i < N_STN; i++ ){

    if ( n_STN -> s[ i ] ){

      fprintf ( n_STN -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_STN -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  //GPe
  for ( int i = 0; i < N_GPe; i++ ){

    if ( n_GPe -> s[ i ] ){

      fprintf ( n_GPe -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_GPe -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }
  */

  //GPi
  for ( int i = 0; i < N_GPi; i++ ){

    if ( n_GPi -> s[ i ] ){

      //fprintf ( n_GPi -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_GPi -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  //CSN
  for ( int i = 0; i < N_CSN; i++ ){

    if ( n_CSN -> s[ i ] ){

      //fprintf ( n_CSN -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_CSN -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  //PTN
  for ( int i = 0; i < N_PTN; i++ ){

    if ( n_PTN -> s[ i ] ){

      //fprintf ( n_PTN -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_PTN -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }

  /*
  //CMPf
  for ( int i = 0; i < N_CMPf; i++ ){

    if ( n_CMPf -> s[ i ] ){

      fprintf ( n_CMPf -> file, "%f %d\n", DT * ( nt + 1 ), i );
      n_CMPf -> counter[ i ]++;
    } // Spike time is not t but t + DT
  }
  */
}