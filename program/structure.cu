#include <stdio.h>
#include <stdlib.h>
#include <SFMT.h>
#include "param.h"

//ニューロン
typedef struct{

  float *v, *ig, *alpha_AMPA, *alpha_NMDA, *alpha_GABA;
  bool *s, *select;
  int *refr, *ts, *counter, *post;
  long  *num_pre;
  sfmt_t rng;
  FILE *file, *file1, *file2;
} neuron_t;

typedef enum{
  MSN,
  FSI,
  STN,
  GPe,
  GPi,
  CSN,
  PTN,
  CMPf
} neuron_kind;

void fileclose( neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN ){

  // fclose ( n_MSN -> file );
  // fclose ( n_MSN -> file1 );
  // fclose ( n_MSN -> file2 );

  // fclose ( n_FSI -> file );
  // fclose ( n_FSI -> file1 );
  // fclose ( n_FSI -> file2 );

  // fclose ( n_STN -> file );
  // fclose ( n_STN -> file1 );
  // fclose ( n_STN -> file2 );

  // fclose ( n_GPe -> file );
  // fclose ( n_GPe -> file1 );
  // fclose ( n_GPe -> file2 );

  // fclose ( n_GPi -> file );
  fclose ( n_GPi -> file1 );
  fclose ( n_GPi -> file2 );

  // fclose ( n_CSN -> file );
  fclose ( n_CSN -> file1 );
  fclose ( n_CSN -> file2 );

  // fclose ( n_PTN -> file );
  fclose ( n_PTN -> file1 );
  fclose ( n_PTN -> file2 );

  // fclose ( n_CMPf -> file );
  // fclose ( n_CMPf -> file1 );
  // fclose ( n_CMPf -> file2 );
}

void finalize_neuron ( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

  // MSN
  hipFree ( n_MSN -> v );
  hipFree ( n_MSN -> ig );
  hipFree ( n_MSN -> alpha_GABA );
  hipFree ( n_MSN -> ts );
  hipFree ( n_MSN -> s );
  hipFree ( n_MSN -> refr );
  //hipFree ( n_MSN -> counter );
  hipFree ( n_MSN -> post );
  hipFree ( n_MSN -> num_pre );

  // FSI
  hipFree ( n_FSI -> v );
  hipFree ( n_FSI -> ig );
  hipFree ( n_FSI -> alpha_GABA );
  hipFree ( n_FSI -> ts );
  hipFree ( n_FSI -> s );
  hipFree ( n_FSI -> refr );
  //hipFree ( n_FSI -> counter );
  hipFree ( n_FSI -> post );
  hipFree ( n_FSI -> num_pre );

  // STN
  hipFree ( n_STN -> v );
  hipFree ( n_STN -> ig );
  hipFree ( n_STN -> alpha_AMPA );
  hipFree ( n_STN -> alpha_NMDA );
  hipFree ( n_STN -> ts );
  hipFree ( n_STN -> s );
  //hipFree ( n_STN -> refr );
  hipFree ( n_STN -> counter );
  hipFree ( n_STN -> post );
  hipFree ( n_STN -> num_pre );

  // GPe
  hipFree ( n_GPe -> v );
  hipFree ( n_GPe -> ig );
  hipFree ( n_GPe -> alpha_GABA );
  hipFree ( n_GPe -> ts );
  hipFree ( n_GPe -> s );
  hipFree ( n_GPe -> refr );
  //hipFree ( n_GPe -> counter );
  hipFree ( n_GPe -> post );
  hipFree ( n_GPe -> num_pre );

  // GPi
  hipFree ( n_GPi -> v );
  hipFree ( n_GPi -> ig );
  hipFree ( n_GPi -> ts );
  hipFree ( n_GPi -> s );
  hipFree ( n_GPi -> refr );
  hipFree ( n_GPi -> counter );
  hipFree ( n_GPi -> post );
  hipFree ( n_GPi -> num_pre );

  // CSN
  hipFree ( n_CSN -> alpha_AMPA );
  hipFree ( n_CSN -> alpha_NMDA );
  hipFree ( n_CSN -> ts );
  hipFree ( n_CSN -> s );
  hipFree ( n_CSN -> refr );
  hipFree ( n_CSN -> counter );

  // PTN
  hipFree ( n_PTN -> alpha_AMPA );
  hipFree ( n_PTN -> alpha_NMDA );
  hipFree ( n_PTN -> ts );
  hipFree ( n_PTN -> s );
  hipFree ( n_PTN -> refr );
  hipFree ( n_PTN -> counter );

  // CMPf
  hipFree ( n_CMPf -> alpha_AMPA );
  hipFree ( n_CMPf -> alpha_NMDA );
  hipFree ( n_CMPf -> ts );
  hipFree ( n_CMPf -> s );
  hipFree ( n_CMPf -> refr );
  //hipFree ( n_CMPf -> counter );
}

void finalize ( neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

  finalize_neuron ( n_MSN, n_FSI, n_STN, n_GPe, n_GPi, n_CSN, n_PTN, n_CMPf );
}