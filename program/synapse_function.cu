#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <SFMT.h>
#include "param.h"

//γ
__device__ float DistanceDecay( int x, int y ){

    float r;
    if ( x == MSN ){

        if ( y == CSN ){

            r = cosh ( L_MSN * ( 1 - p_CSNMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else if ( y == PTN ){

            r = cosh ( L_MSN * ( 1 - p_PTNMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else if ( y == MSN ){

            r = cosh ( L_MSN * ( 1 - p_MSNMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else if ( y == FSI ){

            r = cosh ( L_MSN * ( 1 - p_FSIMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else if ( y == STN ){

            r = cosh ( L_MSN * ( 1 - p_STNMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else if ( y == GPe ){

            r = cosh ( L_MSN * ( 1 - p_GPeMSN ) ) / cosh ( L_MSN );
            return r;
        }
        else {

            r = cosh ( L_MSN * ( 1 - p_CMPfMSN ) ) / cosh ( L_MSN );
            return r;
        }
    }
    else if ( x == FSI ){

        if ( y == CSN ){

            r = cosh ( L_FSI * ( 1 - p_CSNFSI ) ) / cosh ( L_FSI );
            return r;
        }
        else if ( y == PTN ){

            r = cosh ( L_FSI * ( 1 - p_PTNFSI ) ) / cosh ( L_FSI );
            return r;
        }
        else if ( y == FSI ){

            r = cosh ( L_FSI * ( 1 - p_FSIFSI ) ) / cosh ( L_FSI );
            return r;
        }
        else if ( y == STN ){

            r = cosh ( L_FSI * ( 1 - p_STNFSI ) ) / cosh ( L_FSI );
            return r;
        }
        else if ( y == GPe ){

            r = cosh ( L_FSI * ( 1 - p_GPeFSI ) ) / cosh ( L_FSI );
            return r;
        }
        else {

            r = cosh ( L_FSI * ( 1 - p_CMPfFSI ) ) / cosh ( L_FSI );
            return r;
        }
    }
    else if ( x == STN ){

        if ( y == PTN ){

            r = cosh ( L_STN * ( 1 - p_PTNSTN ) ) / cosh ( L_STN );
            return r;
        }
        else if ( y == GPe ){

            r = cosh ( L_STN * ( 1 - p_GPeSTN ) ) / cosh ( L_STN );
            return r;
        }
        else {

            r = cosh ( L_STN * ( 1 - p_CMPfSTN ) ) / cosh ( L_STN );
            return r;
        }
    }
    else if ( x == GPe ){

        if ( y == MSN ){

            r = cosh ( L_GPe * ( 1 - p_MSNGPe ) ) / cosh ( L_GPe );
            return r;
        }
        else if ( y == STN ){

            r = cosh ( L_GPe * ( 1 - p_STNGPe ) ) / cosh ( L_GPe );
            return r;
        }
        else if ( y == GPe ){

            r = cosh ( L_GPe * ( 1 - p_GPeGPe ) ) / cosh ( L_GPe );
            return r;
        }
        else {

            r = cosh ( L_GPe * ( 1 - p_CMPfGPe ) ) / cosh ( L_GPe );
            return r;
        }
    }
    else{

        if ( y == MSN ){

            r = cosh ( L_GPi * ( 1 - p_MSNGPi ) ) / cosh ( L_GPi );
            return r;
        }
        else if ( y == STN ){

            r = cosh ( L_GPi * ( 1 - p_STNGPi ) ) / cosh ( L_GPi );
            return r;
        }
        else if ( y == GPe ){

            r = cosh ( L_GPi * ( 1 - p_GPeGPi ) ) / cosh ( L_GPi );
            return r;
        }
        else {

            r = cosh ( L_GPi * ( 1 - p_CMPfGPi ) ) / cosh ( L_GPi );
            return r;
        }
    }
}

__device__ void ExcSynapse ( int nt, long i, neuron_t *preneuron ){

    preneuron -> alpha_AMPA[ i ] = ( float ) tauAMPA * preneuron -> alpha_AMPA[ i ];
    preneuron -> alpha_NMDA[ i ] = ( float ) tauNMDA * preneuron -> alpha_NMDA[ i ] + W_NMDA * ( preneuron -> ts[ i ] + DELAY == nt );;
}
__device__ void InhSynapse ( int nt, long i, neuron_t *preneuron ){

    preneuron -> alpha_GABA[ i ] = ( float ) tauGABA * preneuron -> alpha_GABA[ i ];
}

__global__ void updateSynapse_MSN ( int nt, neuron_t *n_MSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN ){
        InhSynapse ( nt, i, n_MSN );
    }
}
__global__ void updateSynapse_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){
        InhSynapse ( nt, i, n_FSI );
    }
}
__global__ void updateSynapse_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){
        ExcSynapse ( nt, i, n_STN );
    }
}
__global__ void updateSynapse_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){
        InhSynapse ( nt, i, n_GPe );
    }
}
__global__ void updateSynapse_CSN ( int nt, neuron_t *n_CSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CSN ){
        ExcSynapse ( nt, i, n_CSN );
    }
}
__global__ void updateSynapse_PTN ( int nt, neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_PTN ){
        ExcSynapse ( nt, i, n_PTN );
    }
}
__global__ void updateSynapse_CMPf ( int nt, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CMPf ){
        ExcSynapse ( nt, i, n_CMPf );
    }
}
void updateSynapse ( int nt, neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    updateSynapse_MSN <<< GRID_SIZE_MSN, BLOCK_SIZE >>> ( nt, n_MSN );
    hipDeviceSynchronize ( );
    updateSynapse_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    updateSynapse_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    updateSynapse_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    updateSynapse_CSN <<< GRID_SIZE_CSN, BLOCK_SIZE >>> ( nt, n_CSN );
    hipDeviceSynchronize ( );
    updateSynapse_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN );
    hipDeviceSynchronize ( );
    updateSynapse_CMPf <<< GRID_SIZE_CMPf, BLOCK_SIZE >>> ( nt, n_CMPf );
    hipDeviceSynchronize ( );
}

__global__ void InputSynapsePotential_MSN ( int nt, neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    neuron_kind x, y;

    if ( i < N_MSN ){
        float r = 0;
        int t = 0;
        x = MSN;
        y = MSN;
        for ( int j = 0; j < N_MSN; j++ ){
            r -= ( n_MSN -> alpha_GABA[ j ] + W_MSNMSN * ( n_MSN -> ts[ j ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;

        y = FSI;
        for ( int j = 0; j < ( n_MSN -> num_pre [ i + ( t * N_MSN ) ] - n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ); j++ ){ 
            r -= ( n_FSI -> alpha_GABA [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseFSI ] + W_FSIMSN * ( n_FSI -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseFSI ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;

        y = STN;
        for ( int j = 0; j < ( n_MSN -> num_pre [ i + ( t * N_MSN ) ] - n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ); j++ ){ 
            r += ( ( n_STN -> alpha_AMPA [ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseSTN ] + W_STNMSN * ( n_STN -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseSTN ] + DELAY == nt ) ) + n_STN -> alpha_NMDA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseSTN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = GPe;
        for ( int j = 0; j < ( n_MSN -> num_pre [ i + ( t * N_MSN ) ] - n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ); j++ ){ 
            r -= ( n_GPe -> alpha_GABA [ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseGPe ] + W_GPeMSN * ( n_GPe -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseGPe ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;

        y = CSN;
        for ( int j = 0; j < ( n_MSN -> num_pre[ i + ( t * N_MSN ) ] - n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ); j++ ){ 
            r += ( ( n_CSN -> alpha_AMPA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseCSN ] + W_CSNMSN * ( n_CSN -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseCSN ] + DELAY == nt ) ) + n_CSN -> alpha_NMDA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseCSN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = PTN;
        for ( int j = 0; j < ( n_MSN -> num_pre[ i + ( t * N_MSN ) ] - n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ); j++ ){ 
            r += ( ( n_PTN -> alpha_AMPA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapsePTN ] + W_PTNMSN * ( n_PTN -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapsePTN ] + DELAY == nt ) ) + n_PTN -> alpha_NMDA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapsePTN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( ( n_CMPf -> alpha_AMPA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapseCMPf ] + W_CMPfMSN * ( n_CMPf -> ts [ n_MSN -> post [ j + n_MSN -> num_pre [ i + ( t * N_MSN ) - 1 ] ] - SynapseCMPf ] + DELAY == nt ) ) + n_CMPf -> alpha_NMDA[ n_MSN -> post [ j + n_MSN -> num_pre[ i + ( t * N_MSN ) - 1 ] ] - SynapsePTN  ] ) * DistanceDecay ( x, y );
        }
        n_MSN -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_FSI ( int nt, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){
    
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    neuron_kind x, y;

    if ( i < N_FSI ) {
        float r = 0;
        int t = 0;
        x = FSI;
        y = FSI;
        for ( int j = 0; j < N_FSI; j++ ){ 
            r -= ( n_FSI -> alpha_GABA[ j ] + W_FSIFSI * ( n_FSI -> ts[ j ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;

        y = STN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( ( n_STN -> alpha_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] + W_STNFSI * ( n_STN -> ts [ n_FSI -> post [ j + n_FSI -> num_pre [ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] + DELAY == nt ) ) + n_STN -> alpha_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = GPe;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r -= ( n_GPe -> alpha_GABA[ n_FSI -> post[ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseGPe ]  + W_GPeFSI * ( n_GPe -> ts [ n_FSI -> post [ j + n_FSI -> num_pre [ i + ( t * N_FSI ) - 1 ] ] - SynapseGPe ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;

        y = CSN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( ( n_CSN -> alpha_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseCSN ] + W_CSNFSI * ( n_CSN -> ts [ n_FSI -> post [ j + n_FSI -> num_pre [ i + ( t * N_FSI ) - 1 ] ] - SynapseCSN ] + DELAY == nt ) ) + n_CSN -> alpha_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseCSN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = PTN;
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += ( ( n_PTN -> alpha_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] + W_PTNFSI * ( n_PTN -> ts [ n_FSI -> post [ j + n_FSI -> num_pre [ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] + DELAY == nt ) ) + n_PTN -> alpha_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] ) * DistanceDecay ( x, y );
        }
        t++;

        y = CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( ( n_CMPf -> alpha_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseCMPf ] + W_CMPfFSI * ( n_CMPf -> ts [ n_FSI -> post [ j + n_FSI -> num_pre [ i + ( t * N_FSI ) - 1 ] ] - SynapseCMPf ] + DELAY == nt ) ) + n_CMPf -> alpha_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN  ] ) * DistanceDecay ( x, y );
        }
        n_FSI -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_STN ( int nt, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_PTN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    neuron_kind x, y;

    if ( i < N_STN ){
        float r = 0;
        int t = 0;
        x = STN;
        y = GPe;
        for ( int j = 0; j < N_GPe; j++ ){ 
            r -= ( n_GPe -> alpha_GABA [ j ] + W_GPeSTN * ( n_GPe -> ts [ j ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;
        y = PTN;
        for ( int j = 0; j < ( n_STN -> num_pre[ i + ( t * N_STN ) ] - n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ); j++ ){ 
            r += ( ( n_PTN -> alpha_AMPA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] + W_PTNSTN * ( n_PTN -> ts [ n_STN -> post [ j + n_STN -> num_pre [ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] + DELAY == nt ) ) + n_PTN -> alpha_NMDA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] ) * DistanceDecay ( x, y );
        }
        t++;
        y = CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( ( n_CMPf -> alpha_AMPA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapseCMPf ] + W_CMPfSTN * ( n_CMPf -> ts [ n_STN -> post [ j + n_STN -> num_pre [ i + ( t * N_STN ) - 1 ] ] - SynapseCMPf ] + DELAY == nt ) ) + n_CMPf -> alpha_NMDA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN  ] ) * DistanceDecay ( x, y );
        }
        n_STN -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_GPe ( int nt, neuron_t *n_MSN, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    neuron_kind x, y;

    if ( i < N_GPe ){
        float r = 0;
        int t = 0;
        x = GPe;
        y = MSN;
        for ( int j = 0; j < N_MSN; j++ ){
            r -= ( n_MSN -> alpha_GABA[ j ] + W_MSNGPe * ( n_MSN -> ts[ j ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;
        y = STN;
        for ( int j = 0; j < ( n_GPe -> num_pre [ i + ( t * N_GPe ) ] - n_GPe -> num_pre [ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r += ( ( n_STN -> alpha_AMPA [ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] + W_STNGPe * ( n_STN -> ts [ n_GPe -> post [ j + n_GPe -> num_pre [ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] + DELAY == nt ) ) + n_STN -> alpha_NMDA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] ) * DistanceDecay ( x, y );
        }
        t++;
        y = GPe;
        for ( int j = 0; j < ( n_GPe -> num_pre [ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r -= ( n_GPe -> alpha_GABA [ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseGPe ] + W_GPeGPe * ( n_GPe -> ts [ n_GPe -> post [ j + n_GPe -> num_pre [ i + ( t * N_GPe ) - 1 ] ] - SynapseGPe ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;
        y = CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( ( n_CMPf -> alpha_AMPA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseCMPf ] + W_CMPfGPe * ( n_CMPf -> ts [ n_GPe -> post [ j + n_GPe -> num_pre [ i + ( t * N_GPe ) - 1 ] ] - SynapseCMPf ] + DELAY == nt ) ) + n_CMPf -> alpha_NMDA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapsePTN  ] ) * DistanceDecay ( x, y );
        }
        n_GPe -> ig[ i ] = r;
    }
}
__global__ void InputSynapsePotential_GPi ( int nt, neuron_t *n_MSN, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    neuron_kind x, y;

    if ( i < N_GPi ){
        float r = 0;
        int t = 0;
        x = GPi;
        y = CMPf;
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += ( ( n_CMPf -> alpha_AMPA[ j ] + W_CMPfGPi * ( n_CMPf -> ts [ j ] + DELAY == nt ) ) + n_CMPf -> alpha_NMDA[ j ] ) * DistanceDecay ( x, y );
        }
        t++;
        y = STN;
        for ( int j = 0; j < ( n_GPi -> num_pre [ i + ( t * N_GPi ) ] - n_GPi -> num_pre [ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r += ( ( n_STN -> alpha_AMPA [ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] + W_STNGPi * ( n_STN -> ts [ n_GPi -> post [ j + n_GPi -> num_pre [ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] + DELAY == nt ) ) + n_STN -> alpha_NMDA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] ) * DistanceDecay ( x, y );
        }
        t++;
        y = GPe;
        for ( int j = 0; j < ( n_GPi -> num_pre [ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r -= ( n_GPe -> alpha_GABA [ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseGPe ] + W_GPeGPi * ( n_GPe -> ts [ n_GPi -> post [ j + n_GPi -> num_pre [ i + ( t * N_GPi ) - 1 ] ] - SynapseGPe ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        t++;
        y = MSN;
        for ( int j = 0; j < ( n_GPi -> num_pre [ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){
            r -= ( n_MSN -> alpha_GABA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN ] + W_MSNGPi * ( n_MSN -> ts[ n_GPi -> post [ j + n_GPi -> num_pre [ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN ] + DELAY == nt ) ) * DistanceDecay ( x, y );
        }
        n_GPi -> ig[ i ] = r;
    }
}
void InputSynapsePotential ( int nt, neuron_t *n_MSN , neuron_t *n_FSI , neuron_t *n_STN , neuron_t *n_GPe , neuron_t *n_GPi, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    InputSynapsePotential_MSN <<< GRID_SIZE_MSN, BLOCK_SIZE >>> ( nt, n_MSN, n_FSI, n_STN, n_GPe, n_CSN, n_PTN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI, n_STN, n_GPe, n_CSN, n_PTN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN, n_GPe, n_PTN, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_MSN, n_STN, n_GPe, n_CMPf );
    hipDeviceSynchronize ( );
    InputSynapsePotential_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_MSN, n_STN, n_GPe, n_GPi, n_CMPf );
    hipDeviceSynchronize ( );
}