#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>

//膜電位の更新
// MSN
__global__ void updatePotential_MSN ( int nt, neuron_t *n_MSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN ){

        n_MSN -> v[ i ] += DT * ( - ( n_MSN -> v[ i ] ) + Vc_MSN + ( rho * n_MSN -> ig[ i ] ) ) / TAU_MSN;
        n_MSN -> s[ i ] = ( n_MSN -> v[ i ] > THETA_MSN );
        n_MSN -> ts[ i ] = ( n_MSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_MSN -> s[ i ] ) ) * ( n_MSN -> ts[ i ] );
        n_MSN -> v[ i ] = ( n_MSN -> s[ i ] ) * V_RESET + ( !( n_MSN -> s[ i ] ) ) * n_MSN -> v[ i ];
        n_MSN -> ig[ i ] = V_RESET;
    }
}
// FSI
__global__ void updatePotential_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> v[ i ] += DT * ( - ( n_FSI -> v[ i ] ) + Vc_FSI + ( rho * n_FSI -> ig[ i ] ) ) / TAU_FSI;
        n_FSI -> s[ i ] = ( n_FSI -> v[ i ] > THETA_FSI );
        n_FSI -> ts[ i ] = ( n_FSI -> s[ i ] ) * ( nt + 1 ) + ( !( n_FSI -> s[ i ] ) ) * ( n_FSI -> ts[ i ] );
        n_FSI -> v[ i ] = ( n_FSI -> s[ i ] ) * V_RESET + ( !( n_FSI -> s[ i ] ) ) * n_FSI -> v[ i ];
        n_FSI -> ig[ i ] = V_RESET;
    }
}
// STN
__global__ void updatePotential_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> v[ i ] += DT * ( - ( n_STN -> v[ i ] ) + Vc_STN + ( rho * n_STN -> ig[ i ] ) ) / TAU_STN;
        n_STN -> s[ i ] = ( n_STN -> v[ i ] > THETA_STN );
        n_STN -> ts[ i ] = ( n_STN -> s[ i ]) * ( nt + 1 ) + ( !( n_STN -> s[ i ] ) ) * ( n_STN -> ts[ i ] );
        n_STN -> v[ i ] = ( n_STN -> s[ i ] ) * V_RESET + ( !( n_STN -> s[ i ] ) ) * n_STN -> v[ i ];
        n_STN -> ig[ i ] = V_RESET;
    }
}
// GPe
__global__ void updatePotential_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> v[ i ] += DT * ( - ( n_GPe -> v[ i ] ) + Vc_GPe + ( rho * n_GPe -> ig[ i ] ) ) / TAU_GPe;
        n_GPe -> s[ i ] = ( n_GPe -> v[ i ] > THETA_GPe );
        n_GPe -> ts[ i ] = ( n_GPe -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPe -> s[ i ] ) ) * ( n_GPe -> ts[ i ] );
        n_GPe -> v[ i ] = ( n_GPe -> s[ i ] ) * V_RESET + ( !( n_GPe -> s[ i ] ) ) * n_GPe -> v[ i ];
        n_GPe -> ig[ i ] = V_RESET;
    }
}
// GPi
__global__ void updatePotential_GPi ( int nt, neuron_t *n_GPi ){
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> v[ i ] += DT * ( - ( n_GPi -> v[ i ] ) + Vc_GPi + ( rho * n_GPi -> ig[ i ] ) ) / TAU_GPi;
        n_GPi -> s[ i ] = ( n_GPi -> v[ i ] > THETA_GPi );
        n_GPi -> ts[ i ] = ( n_GPi -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPi -> s[ i ] ) ) * ( n_GPi -> ts[ i ] );
        n_GPi -> v[ i ] = ( n_GPi -> s[ i ] ) * V_RESET + ( !( n_GPi -> s[ i ] ) ) * n_GPi -> v[ i ];
        n_GPi -> ig[ i ] = V_RESET;
    }
}

void updatePotential ( int nt, neuron_t *n_MSN, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi ){

    updatePotential_MSN <<< GRID_SIZE_MSN, BLOCK_SIZE >>> ( nt, n_MSN );
    hipDeviceSynchronize ( );
    updatePotential_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    updatePotential_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    updatePotential_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    updatePotential_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_GPi );
    hipDeviceSynchronize ( );
}

//ポアソンモデル
void change_inputneuron ( int nt, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf, int w, int z ){

    int i;
    //CSN
    for ( i = 0; i < per_chanel_N_CSN; i++ ){
        double r = sfmt_genrand_real2 ( &( n_CSN -> rng ) );
        n_CSN -> s [ i ] = r < ( PHI_MIN_CSN + ( ( ( ( PHI_MAX_CSN - PHI_MIN_CSN ) * w ) / N_i ) * n_CSN -> select [ i ] ) );
        n_CSN -> ts[ i ] = ( n_CSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_CSN -> s[ i ] ) ) * ( n_CSN -> ts[ i ] );
    }
    for ( ; i < N_CSN; i++ ){
        double r = sfmt_genrand_real2 ( &( n_CSN -> rng ) );
        n_CSN -> s [ i ] = r < ( PHI_MIN_CSN + ( ( ( ( PHI_MAX_CSN - PHI_MIN_CSN ) * z ) / N_i ) * n_CSN -> select [ i ] ) );
        n_CSN -> ts[ i ] = ( n_CSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_CSN -> s[ i ] ) ) * ( n_CSN -> ts[ i ] );
    }

    //PTN
    for ( i = 0; i < per_chanel_N_PTN; i++ ){
        double r = sfmt_genrand_real2 ( &( n_PTN -> rng ) );
        n_PTN -> s [ i ] =  r < ( PHI_MIN_PTN + ( ( ( PHI_MAX_PTN - PHI_MIN_PTN ) * w ) / N_i ) * n_PTN -> select [ i ] );
        n_PTN -> ts[ i ] = ( n_PTN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTN -> s[ i ] ) ) * ( n_PTN -> ts[ i ] );
    }
    for ( ; i < N_PTN; i++ ){
        double r = sfmt_genrand_real2 ( &( n_PTN -> rng ) );
        n_PTN -> s [ i ] =  r < ( PHI_MIN_PTN + ( ( ( PHI_MAX_PTN - PHI_MIN_PTN ) * z ) / N_i ) * n_PTN -> select [ i ] );
        n_PTN -> ts[ i ] = ( n_PTN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTN -> s[ i ] ) ) * ( n_PTN -> ts[ i ] );
    }

    //CMPf
    for ( int i = 0; i < N_CMPf; i++ ){
        double r = sfmt_genrand_real2 ( &( n_CMPf -> rng ) );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}

void inputneuron ( int nt, neuron_t *n_CSN, neuron_t *n_PTN, neuron_t *n_CMPf ){

    //CSN
    for ( int i = 0; i < N_CSN; i++ ){

        double r = sfmt_genrand_real2 ( &( n_CSN -> rng ) );
        n_CSN -> s[ i ] = ( r < PHI_MIN_CSN );
        n_CSN -> ts[ i ] = ( n_CSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_CSN -> s[ i ] ) ) * ( n_CSN -> ts[ i ] );
    }

    //PTN
    for ( int i = 0; i < N_PTN; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PTN -> rng ) );
        n_PTN -> s[ i ] = ( r < PHI_MIN_PTN );
        n_PTN -> ts[ i ] = ( n_PTN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTN -> s[ i ] ) ) * ( n_PTN -> ts[ i ] );
    }

    //CMPf
    for ( int i = 0; i < N_CMPf; i++ ){

        double r = sfmt_genrand_real2 ( &( n_CMPf -> rng ) );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}