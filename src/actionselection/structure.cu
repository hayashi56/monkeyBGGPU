#include <stdio.h>
#include <stdlib.h>
#include <SFMT.h>
#include "param.h"

// ニューロンに関する値を定義している構造体
/*
vはニューロンの膜電位
igはニューロンが受け取るシナプス後電位
alpha_〇〇はニューロンからの神経伝達物質によって生じるシナプス後電位(GABAは抑制性、AMPA, NMDAは興奮性、DOPAは受容体によってドーパミンが興奮性、抑制性に変化)
sはニューロンの発火の有無
selectは行動選択を行う際に発火率を変化させるニューロンであるかどうか
refrは不応期の残りカウント(2msの不応期であれば2)
tsは発火した時間
counterは発火したニューロンの数のカウント
postはそのニューロンがどのニューロンとの間に結合を持ているかを相手のニューロン番号を入れることで記録
num_preはそのニューロンがそれぞれの種類のニューロンに対していくつの結合を持っているか
rngはニューロンの初期化や入力のニューロンの発火の有無を求めるときに扱う乱数のシード値(乱数生成方法はメルセンヌ・ツイスタ)
fileはシミュレーション内の時間においてそのニューロンの発火を記述する出力ファイル(具体的には発火した時間と発火したニューロンの番号の出力)
*/
typedef struct{

    float *v, *ig, *psp_AMPA, *psp_GABA, *psp_NMDA, *psp_DOPA;
    bool *s, *select;
    int *refr, *ts, *counter, *post;
    long *num_pre;
    sfmt_t rng;
    FILE *file;
} neuron_t;

// それぞれニューロンのデータを出力したファイルを閉じる関数
void fileclose( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    fclose ( n_MSN_D1 -> file );
    fclose ( n_MSN_D2 -> file );
    fclose ( n_FSI -> file );
    fclose ( n_STN -> file );
    fclose ( n_GPe -> file );
    fclose ( n_GPi -> file );
    fclose ( n_SNc -> file );
    fclose ( n_PTI -> file );
    fclose ( n_PTN -> file );
    fclose ( n_PSN -> file );
    fclose ( n_Th -> file );
    fclose ( n_CMPf -> file );
}

// それぞれの構造体内のニューロンに関す情報を持ったポインタ変数のメモリの開放
void finalize ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    // MSN_D1
    hipFree ( n_MSN_D1 -> v );
    hipFree ( n_MSN_D1 -> ig );
    hipFree ( n_MSN_D1 -> psp_GABA );
    hipFree ( n_MSN_D1 -> ts );
    hipFree ( n_MSN_D1 -> s );
    hipFree ( n_MSN_D1 -> refr );
    hipFree ( n_MSN_D1 -> counter );
    hipFree ( n_MSN_D1 -> post );
    hipFree ( n_MSN_D1 -> num_pre );

    // MSN_D2
    hipFree ( n_MSN_D2 -> v );
    hipFree ( n_MSN_D2 -> ig );
    hipFree ( n_MSN_D2 -> psp_GABA );
    hipFree ( n_MSN_D2 -> ts );
    hipFree ( n_MSN_D2 -> s );
    hipFree ( n_MSN_D2 -> refr );
    hipFree ( n_MSN_D2 -> counter );
    hipFree ( n_MSN_D2 -> post );
    hipFree ( n_MSN_D2 -> num_pre );

    // FSI
    hipFree ( n_FSI -> v );
    hipFree ( n_FSI -> ig );
    hipFree ( n_FSI -> psp_GABA );
    hipFree ( n_FSI -> ts );
    hipFree ( n_FSI -> s );
    hipFree ( n_FSI -> refr );
    hipFree ( n_FSI -> counter );
    hipFree ( n_FSI -> post );
    hipFree ( n_FSI -> num_pre );

    // STN
    hipFree ( n_STN -> v );
    hipFree ( n_STN -> ig );
    hipFree ( n_STN -> psp_AMPA );
    hipFree ( n_STN -> psp_NMDA );
    hipFree ( n_STN -> ts );
    hipFree ( n_STN -> s );
    hipFree ( n_STN -> refr );
    hipFree ( n_STN -> counter );
    hipFree ( n_STN -> post );
    hipFree ( n_STN -> num_pre );

    // GPe
    hipFree ( n_GPe -> v );
    hipFree ( n_GPe -> ig );
    hipFree ( n_GPe -> psp_GABA );
    hipFree ( n_GPe -> ts );
    hipFree ( n_GPe -> s );
    hipFree ( n_GPe -> refr );
    hipFree ( n_GPe -> counter );
    hipFree ( n_GPe -> post );
    hipFree ( n_GPe -> num_pre );

    // GPi
    hipFree ( n_GPi -> v );
    hipFree ( n_GPi -> ig );
    hipFree ( n_GPi -> psp_GABA );
    hipFree ( n_GPi -> ts );
    hipFree ( n_GPi -> s );
    hipFree ( n_GPi -> refr );
    hipFree ( n_GPi -> counter );
    hipFree ( n_GPi -> post );
    hipFree ( n_GPi -> num_pre );

    // SNc
    hipFree ( n_SNc -> v );
    hipFree ( n_SNc -> ig );
    hipFree ( n_SNc -> psp_DOPA );
    hipFree ( n_SNc -> ts );
    hipFree ( n_SNc -> s );
    hipFree ( n_SNc -> refr );
    hipFree ( n_SNc -> counter );
    hipFree ( n_SNc -> post );
    hipFree ( n_SNc -> num_pre );

    // PTN
    hipFree ( n_PTN -> v );
    hipFree ( n_PTN -> ig );
    hipFree ( n_PTN -> psp_AMPA );
    hipFree ( n_PTN -> psp_NMDA );
    hipFree ( n_PTN -> ts );
    hipFree ( n_PTN -> s );
    hipFree ( n_PTN -> refr );
    hipFree ( n_PTN -> counter );
    hipFree ( n_PTN -> post );
    hipFree ( n_PTN -> num_pre );

    // PTI
    hipFree ( n_PTI -> v );
    hipFree ( n_PTI -> ig );
    hipFree ( n_PTI -> psp_GABA );
    hipFree ( n_PTI -> ts );
    hipFree ( n_PTI -> s );
    hipFree ( n_PTI -> refr );
    hipFree ( n_PTI -> counter );
    hipFree ( n_PTI -> post );
    hipFree ( n_PTI -> num_pre );

    // PSN
    hipFree ( n_PSN -> psp_AMPA );
    hipFree ( n_PSN -> psp_NMDA );
    hipFree ( n_PSN -> ts );
    hipFree ( n_PSN -> s );
    hipFree ( n_PSN -> counter );
    hipFree ( n_PSN -> select );

    // Th
    hipFree ( n_Th -> v );
    hipFree ( n_Th -> ig );
    hipFree ( n_Th -> psp_AMPA );
    hipFree ( n_Th -> psp_NMDA );
    hipFree ( n_Th -> ts );
    hipFree ( n_Th -> s );
    hipFree ( n_Th -> refr );
    hipFree ( n_Th -> counter );
    hipFree ( n_Th -> post );
    hipFree ( n_Th -> num_pre );

    // CMPf
    hipFree ( n_CMPf -> psp_AMPA );
    hipFree ( n_CMPf -> psp_NMDA );
    hipFree ( n_CMPf -> ts );
    hipFree ( n_CMPf -> s );
    hipFree ( n_CMPf -> counter );
}