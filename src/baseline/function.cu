#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include <time.h>

extern "C" { void timer_start( void ); }
extern "C" { double timer_elapsed( void ); }

// 積分発火型モデルで再現するそれぞれのニューロンの不応期を再現している関数
__global__ void t_refr_MSN_D1 ( neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){

        n_MSN_D1 -> refr [ i ] = ( n_MSN_D1 -> s [ i ] ) * ( T_REFR ) + ( !( n_MSN_D1 -> s [ i ] ) ) * ( ( n_MSN_D1 -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_MSN_D2 ( neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){

        n_MSN_D2 -> refr [ i ] = ( n_MSN_D2 -> s [ i ] ) * ( T_REFR ) + ( !( n_MSN_D2 -> s [ i ] ) ) * ( ( n_MSN_D2 -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_FSI ( neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> refr [ i ] = ( n_FSI -> s [ i ] ) * ( T_REFR ) + ( !( n_FSI -> s [ i ] ) ) * ( ( n_FSI -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_STN ( neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> refr [ i ] = ( n_STN -> s [ i ] ) * ( T_REFR ) + ( !( n_STN -> s [ i ] ) ) * ( ( n_STN -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_GPe ( neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> refr [ i ] = ( n_GPe -> s [ i ] ) * ( T_REFR ) + ( !( n_GPe -> s [ i ] ) ) * ( ( n_GPe -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_GPi ( neuron_t *n_GPi ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> refr [ i ] = ( n_GPi -> s [ i ] ) * ( T_REFR ) + ( !( n_GPi -> s [ i ] ) ) * ( ( n_GPi -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_SNc ( neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){

        n_SNc -> refr [ i ] = ( n_SNc -> s [ i ] ) * ( T_REFR ) + ( !( n_SNc -> s [ i ] ) ) * ( ( n_SNc -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_PTN ( neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTN ){

        n_PTN -> refr [ i ] = ( n_PTN -> s [ i ] ) * ( T_REFR ) + ( !( n_PTN -> s [ i ] ) ) * ( ( n_PTN -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_PTI ( neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){

        n_PTI -> refr [ i ] = ( n_PTI -> s [ i ] ) * ( T_REFR ) + ( !( n_PTI -> s [ i ] ) ) * ( ( n_PTI -> refr [ i ] ) - 1 ); // set counter
    }
}
__global__ void t_refr_Th ( neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_Th ){

        n_Th -> refr [ i ] = ( n_Th -> s [ i ] ) * ( T_REFR ) + ( !( n_Th -> s [ i ] ) ) * ( ( n_Th -> refr [ i ] ) - 1 ); // set counter
    }
}

// 不応期を再現している関数をグリッド数とブロック数を指定してGPUで実行する関数
void t_refr ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_Th ){

    t_refr_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( n_MSN_D1 );
    hipDeviceSynchronize ( );
    t_refr_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( n_MSN_D2 );
    hipDeviceSynchronize ( );
    t_refr_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( n_FSI );
    hipDeviceSynchronize ( );
    t_refr_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( n_STN );
    hipDeviceSynchronize ( );
    t_refr_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( n_GPe );
    hipDeviceSynchronize ( );
    t_refr_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( n_GPi );
    hipDeviceSynchronize ( );
    t_refr_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( n_SNc );
    hipDeviceSynchronize ( );
    t_refr_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( n_PTN );
    hipDeviceSynchronize ( );
    t_refr_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( n_PTI );
    hipDeviceSynchronize ( );
    t_refr_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( n_Th );
    hipDeviceSynchronize ( );
}

// ループによって時間変化を表現しシミュレーションを実行する関数
void simulat( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    // シミュレーションの計算時間の計測を開始
    timer_start();

    // シミュレーション内の時間を表す変数
    int nt;

    // 初期化時のランダム性から影響を受けないようにするための100[ms]のフリーラン
    for ( nt = 0; nt < FreeRun; nt++ ){
        // preニューロン毎のシナプス後電圧を計算
        updateSynapse ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

        // それぞれのpostニューロンが受け取るシナプス後電圧の合計を計算
        InputSynapsePotential ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

        // それぞれのニューロンの膜電位を計算し発火の有無を判定
        updatePotential( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );

        // 入力となるニューロンの発火の有無を判定
        input_neuron ( nt, n_PSN, n_CMPf );

        // 前の関数でわかった発火の有無をもとに不応期を計算
        t_refr ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
    }

    // 行動選択を行うという入力が無い時間範囲を再現するためのループ(0[ms]~1000[ms]の間)
    for ( ; nt < NT; nt++ ){

        // preニューロン毎のシナプス後電圧を計算
        updateSynapse ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

        // それぞれのpostニューロンが受け取るシナプス後電圧の合計を計算
        InputSynapsePotential ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

        // それぞれのニューロンの膜電位を計算し発火の有無を判定
        updatePotential( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );

        // 入力となるニューロンの発火の有無を判定
        input_neuron ( nt, n_PSN, n_CMPf );

        // 前の関数でわかった発火の有無をもとに不応期を計算
        t_refr ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );

        // それぞれのニューロンについて発火しているニューロンをそのシミュレーション内の時間と共に出力
        outputSpike ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
    }

    // シミュレーションの計算時間
    double elapsedTime = timer_elapsed ();

    // シミュレーションに要した計算時間の出力
    printf ( "Elapsed time = %f sec.\n", elapsedTime );

    // 1000msのシミュレーションでのそれぞれのニューロンの１個あたりの発火率をファイル出力
    outputFiringRate ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
}