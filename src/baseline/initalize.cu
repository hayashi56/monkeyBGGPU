#include <stdio.h>
#include <stdlib.h>
#include <SFMT.h>
#include <random>
#include <algorithm>
#include <vector>
#include "param.h"

// それぞれのニューロンのデータを出力するためのファイルを参照する関数
void file_open ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    n_MSN_D1 -> file = fopen ( "MSN_D1spike.dat", "w" );
    n_MSN_D2 -> file = fopen ( "MSN_D2spike.dat", "w" );
    n_FSI -> file = fopen ( "FSIspike.dat", "w" );
    n_STN -> file = fopen ( "STNspike.dat", "w" );
    n_GPe -> file = fopen ( "GPespike.dat", "w" );
    n_GPi -> file = fopen ( "GPispike.dat", "w" );
    n_SNc -> file = fopen ( "SNcspike.dat", "w" );
    n_PTN -> file = fopen ( "PTNspike.dat", "w" );
    n_PTI -> file = fopen ( "PTIspike.dat", "w" );
    n_PSN -> file = fopen ( "PSNspike.dat", "w" );
    n_Th -> file = fopen ( "Thspike.dat", "w" );
    n_CMPf -> file = fopen ( "CMPfspike.dat", "w" );
}

// 構造体内のそれぞれのニューロンに関する値を格納する配列のメモリ確保をする関数
void Allocating_Neuron ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    hipMallocManaged ( &n_MSN_D1 -> v, sizeof ( float ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> ig, sizeof ( float ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> psp_GABA, sizeof ( float ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> refr, sizeof ( int ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> ts, sizeof ( int ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> s, sizeof ( bool ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> counter, sizeof ( int ) * N_MSN_D1 );
    hipMallocManaged ( &n_MSN_D1 -> num_pre, sizeof ( long ) * N_MSN_D1 * 9 );

    hipMallocManaged ( &n_MSN_D2 -> v, sizeof ( float ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> ig, sizeof ( float ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> psp_GABA, sizeof ( float ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> refr, sizeof ( int ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> ts, sizeof ( int ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> s, sizeof ( bool ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> counter, sizeof ( int ) * N_MSN_D2 );
    hipMallocManaged ( &n_MSN_D2 -> num_pre, sizeof ( long ) * N_MSN_D2 * 9 );

    hipMallocManaged ( &n_FSI -> v, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> ig, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> psp_GABA, sizeof ( float ) * N_FSI );
    hipMallocManaged ( &n_FSI -> s, sizeof ( bool ) * N_FSI );
    hipMallocManaged ( &n_FSI -> ts, sizeof ( int ) * N_FSI );
    hipMallocManaged ( &n_FSI -> refr, sizeof ( int ) * N_FSI );
    hipMallocManaged ( &n_FSI -> counter, sizeof ( int ) * N_FSI );
    hipMallocManaged ( &n_FSI -> num_pre, sizeof ( long ) * N_FSI * 6 );

    hipMallocManaged ( &n_STN -> v, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> ig, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> psp_AMPA, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> psp_NMDA, sizeof ( float ) * N_STN );
    hipMallocManaged ( &n_STN -> s, sizeof ( bool ) * N_STN );
    hipMallocManaged ( &n_STN -> ts, sizeof ( int ) * N_STN );
    hipMallocManaged ( &n_STN -> refr, sizeof ( int ) * N_STN );
    hipMallocManaged ( &n_STN -> counter, sizeof ( int ) * N_STN );
    hipMallocManaged ( &n_STN -> num_pre, sizeof ( long ) * N_STN * 3 );

    hipMallocManaged ( &n_GPe -> v, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> ig, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> psp_GABA, sizeof ( float ) * N_GPe );
    hipMallocManaged ( &n_GPe -> s, sizeof ( bool ) * N_GPe );
    hipMallocManaged ( &n_GPe -> ts, sizeof ( int ) * N_GPe );
    hipMallocManaged ( &n_GPe -> refr, sizeof ( int ) * N_GPe );
    hipMallocManaged ( &n_GPe -> counter, sizeof ( int ) * N_GPe );
    hipMallocManaged ( &n_GPe -> num_pre, sizeof ( long ) * N_GPe * 5 );

    hipMallocManaged ( &n_GPi -> v, sizeof ( float ) * N_GPi );
    hipMallocManaged ( &n_GPi -> ig, sizeof ( float ) * N_GPi );
    hipMallocManaged ( &n_GPi -> psp_GABA, sizeof ( float ) * N_GPi );
    hipMallocManaged ( &n_GPi -> s, sizeof ( bool ) * N_GPi );
    hipMallocManaged ( &n_GPi -> ts, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> refr, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> counter, sizeof ( int ) * N_GPi );
    hipMallocManaged ( &n_GPi -> num_pre, sizeof ( long ) * N_GPi * 5 );

    hipMallocManaged ( &n_SNc -> v, sizeof ( float ) * N_SNc );
    hipMallocManaged ( &n_SNc -> ig, sizeof ( float ) * N_SNc );
    hipMallocManaged ( &n_SNc -> psp_DOPA, sizeof ( float ) * N_SNc );
    hipMallocManaged ( &n_SNc -> s, sizeof ( bool ) * N_SNc );
    hipMallocManaged ( &n_SNc -> ts, sizeof ( int ) * N_SNc );
    hipMallocManaged ( &n_SNc -> refr, sizeof ( int ) * N_SNc );
    hipMallocManaged ( &n_SNc -> counter, sizeof ( int ) * N_SNc );
    hipMallocManaged ( &n_SNc -> num_pre, sizeof ( long ) * N_SNc * 2 );

    hipMallocManaged ( &n_PTN -> v, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> ig, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> psp_AMPA, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> psp_NMDA, sizeof ( float ) * N_PTN );
    hipMallocManaged ( &n_PTN -> s, sizeof ( bool ) * N_PTN );
    hipMallocManaged ( &n_PTN -> ts, sizeof ( int ) * N_PTN );
    hipMallocManaged ( &n_PTN -> refr, sizeof ( int ) * N_PTN );
    hipMallocManaged ( &n_PTN -> counter, sizeof ( int ) * N_PTN );
    hipMallocManaged ( &n_PTN -> num_pre, sizeof ( long ) * N_PTN * 3 );

    hipMallocManaged ( &n_PTI -> v, sizeof ( float ) * N_PTI );
    hipMallocManaged ( &n_PTI -> ig, sizeof ( float ) * N_PTI );
    hipMallocManaged ( &n_PTI -> psp_GABA, sizeof ( float ) * N_PTI );
    hipMallocManaged ( &n_PTI -> s, sizeof ( bool ) * N_PTI );
    hipMallocManaged ( &n_PTI -> ts, sizeof ( int ) * N_PTI );
    hipMallocManaged ( &n_PTI -> refr, sizeof ( int ) * N_PTI );
    hipMallocManaged ( &n_PTI -> counter, sizeof ( int ) * N_PTI );
    hipMallocManaged ( &n_PTI -> num_pre, sizeof ( long ) * N_PTI );

    hipMallocManaged ( &n_PSN -> psp_AMPA, sizeof ( float ) * N_PSN );
    hipMallocManaged ( &n_PSN -> psp_NMDA, sizeof ( float ) * N_PSN );
    hipMallocManaged ( &n_PSN -> s, sizeof ( bool ) * N_PSN );
    hipMallocManaged ( &n_PSN -> ts, sizeof ( int ) * N_PSN );
    hipMallocManaged ( &n_PSN -> counter, sizeof ( int ) * N_PSN );
    hipMallocManaged ( &n_PSN -> select, sizeof ( bool ) * N_PSN );

    hipMallocManaged ( &n_Th -> v, sizeof ( float ) * N_Th );
    hipMallocManaged ( &n_Th -> ig, sizeof ( float ) * N_Th );
    hipMallocManaged ( &n_Th -> psp_AMPA, sizeof ( float ) * N_Th );
    hipMallocManaged ( &n_Th -> psp_NMDA, sizeof ( float ) * N_Th );
    hipMallocManaged ( &n_Th -> s, sizeof ( bool ) * N_Th );
    hipMallocManaged ( &n_Th -> ts, sizeof ( int ) * N_Th );
    hipMallocManaged ( &n_Th -> refr, sizeof ( int ) * N_Th );
    hipMallocManaged ( &n_Th -> counter, sizeof ( int ) * N_Th );
    hipMallocManaged ( &n_Th -> num_pre, sizeof ( long ) * N_Th * 2 );

    hipMallocManaged ( &n_CMPf -> psp_AMPA, sizeof ( float ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> psp_NMDA, sizeof ( float ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> s, sizeof ( bool ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> ts, sizeof ( int ) * N_CMPf );
    hipMallocManaged ( &n_CMPf -> counter, sizeof ( int ) * N_CMPf );
}

// ニューロンの値の初期化をする関数
void initalize_Neuron ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    for ( int i = 0; i < N_MSN_D1; i++ ){

        n_MSN_D1 -> v[ i ] = Er_MSN_D1 + 10. * sfmt_genrand_real2 ( &n_MSN_D1 -> rng );
        n_MSN_D1 -> ig[ i ] = 0;
        n_MSN_D1 -> psp_GABA[ i ] = 0;
        n_MSN_D1 -> s[ i ] = false;
        n_MSN_D1 -> ts[ i ] = 1000;
        n_MSN_D1 -> refr[ i ] = 0;
        n_MSN_D1 -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_MSN_D2; i++ ){

        n_MSN_D2 -> v[ i ] = Er_MSN_D2 + 10. * sfmt_genrand_real2 ( &n_MSN_D2 -> rng );
        n_MSN_D2 -> ig[ i ] = 0;
        n_MSN_D2 -> psp_GABA[ i ] = 0;
        n_MSN_D2 -> s[ i ] = false;
        n_MSN_D2 -> ts[ i ] = 1000;
        n_MSN_D2 -> refr[ i ] = 0;
        n_MSN_D2 -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_FSI; i++ ){

        n_FSI -> v[ i ] = Er_FSI + 10. * sfmt_genrand_real2 ( &n_FSI -> rng );
        n_FSI -> ig[ i ] = 0;
        n_FSI -> psp_GABA[ i ] = 0;
        n_FSI -> s[ i ] = false;
        n_FSI -> ts[ i ] = 1000;
        n_FSI -> refr[ i ] = 0;
        n_FSI -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_STN; i++ ){

        n_STN -> v[ i ] = Er_STN + 10. * sfmt_genrand_real2 ( &n_STN -> rng );
        n_STN -> ig[ i ] = 0;
        n_STN -> psp_AMPA[ i ] = 0;
        n_STN -> psp_NMDA[ i ] = 0;
        n_STN -> s[ i ] = false;
        n_STN -> ts[ i ] = 1000;
        n_STN -> refr[ i ] = 0;
        n_STN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_GPe; i++ ){

        n_GPe -> v[ i ] = Er_GPe + 10. * sfmt_genrand_real2 ( &n_GPe -> rng );
        n_GPe -> ig[ i ] = 0;
        n_GPe -> psp_GABA[ i ] = 0;
        n_GPe -> s[ i ] = false;
        n_GPe -> ts[ i ] = 1000;
        n_GPe -> refr[ i ] = 0;
        n_GPe -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_GPi; i++ ){

        n_GPi -> v[ i ] = Er_GPi + 10. * sfmt_genrand_real2 ( &n_GPi -> rng );
        n_GPi -> ig[ i ] = 0;
        n_GPi -> psp_GABA[ i ] = 0;
        n_GPi -> s[ i ] = false;
        n_GPi -> ts[ i ] = 1000;
        n_GPi -> refr[ i ] = 0;
        n_GPi -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_SNc; i++ ){

        n_SNc -> v[ i ] = Er_SNc + 10. * sfmt_genrand_real2 ( &n_SNc -> rng );
        n_SNc -> ig[ i ] = 0;
        n_SNc -> psp_DOPA[ i ] = 0;
        n_SNc -> s[ i ] = false;
        n_SNc -> ts[ i ] = 1000;
        n_SNc -> refr[ i ] = 0;
        n_SNc -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_PTN; i++ ){

        n_PTN -> v[ i ] = Er_PTN + 10. * sfmt_genrand_real2 ( &n_PTN -> rng );
        n_PTN -> ig[ i ] = 0;
        n_PTN -> psp_AMPA[ i ] = 0;
        n_PTN -> psp_NMDA[ i ] = 0;
        n_PTN -> s[ i ] = false;
        n_PTN -> ts[ i ] = 1000;
        n_PTN -> refr[ i ] = 0;
        n_PTN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_PSN; i++ ){

        n_PSN -> psp_AMPA[ i ] = 0;
        n_PSN -> psp_NMDA[ i ] = 0;
        n_PSN -> s[ i ] = false;
        n_PSN -> ts[ i ] = 1000;
        n_PSN -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_PTI; i++ ){

        n_PTI -> v[ i ] = Er_PTI + 10. * sfmt_genrand_real2 ( &n_PTI -> rng );
        n_PTI -> ig[ i ] = 0;
        n_PTI -> psp_GABA[ i ] = 0;
        n_PTI -> s[ i ] = false;
        n_PTI -> ts[ i ] = 1000;
        n_PTI -> refr[ i ] = 0;
        n_PTI -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_Th; i++ ){

        n_Th -> v[ i ] = Er_Th + 10. * sfmt_genrand_real2 ( &n_Th -> rng );
        n_Th -> ig[ i ] = 0;
        n_Th -> psp_AMPA[ i ] = 0;
        n_Th -> psp_NMDA[ i ] = 0;
        n_Th -> s[ i ] = false;
        n_Th -> ts[ i ] = 1000;
        n_Th -> refr[ i ] = 0;
        n_Th -> counter[ i ] = 0;
    }

    for ( int i = 0; i < N_CMPf; i++ ){

        n_CMPf -> psp_AMPA[ i ] = 0;
        n_CMPf -> psp_NMDA[ i ] = 0;
        n_CMPf -> s[ i ] = false;
        n_CMPf -> ts[ i ] = 1000;
        n_CMPf -> counter[ i ] = 0;
    }
}

// シナプスについてのメモリ確保・初期化をする関数
void initalize_synapse( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_Th ){

    int k = 0;
    long post_synapse = 0;
    long count = 0;
    long count_pre = 0;

    srand ( 1 );
    // CMPf→MSN_D1(拡散型)
    post_synapse += N_CMPf * N_MSN_D1;

    // MSN_D1→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND1MSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // MSN_D2→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND2MSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // FSI→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_FSIMSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // STN→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNMSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // GPe→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand ( ) / RAND_MAX ) < P_GPeMSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // SNc→MSN_D1(拡散型？)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_SNc; j++ ){
            count += ( long ) ( ( ( double ) rand ( ) / RAND_MAX ) < P_SNcMSND1 );
        }
    }
    post_synapse += count;
    count = 0;

    // PTN→MSN_D1(集中型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_MSN_D1 * N_chanel;

    // PSN→MSN_D1(集中型)
    post_synapse += per_chanel_N_PSN * per_chanel_N_MSN_D1 * N_chanel;

    hipMallocManaged ( &n_MSN_D1 -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→MSN_D2(拡散型)
    post_synapse += N_CMPf * N_MSN_D2;

    // MSN_D1→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND1MSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // MSN_D2→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND2MSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // FSI→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_FSIMSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // STN→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNMSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // GPe→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand ( ) / RAND_MAX ) < P_GPeMSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // SNc→MSN_D2(拡散型？)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_SNc; j++ ){
            count += ( long ) ( ( ( double ) rand ( ) / RAND_MAX ) < P_SNcMSND2 );
        }
    }
    post_synapse += count;
    count = 0;

    // PTN→MSN_D2(集中型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_MSN_D2 * N_chanel;

    // PSN→MSN_D2(集中型)
    post_synapse += per_chanel_N_PSN * per_chanel_N_MSN_D2 * N_chanel;

    hipMallocManaged ( &n_MSN_D2 -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→FSI(拡散型)
    post_synapse += N_CMPf * N_FSI;

    // FSI→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_FSIFSI );
        }
    }
    post_synapse += count;
    count = 0;

    // STN→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNFSI );
        }
    }
    post_synapse += count;
    count = 0;

    // GPe→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeFSI );
        }
    }
    post_synapse += count;
    count = 0;

    // PTN→FSI(集中型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_FSI * N_chanel;

    // PSN→FSI(集中型)
    post_synapse += per_chanel_N_PSN * per_chanel_N_FSI * N_chanel;

    hipMallocManaged ( &n_FSI -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→STN(拡散型)
    post_synapse += N_CMPf * N_STN;

    // GPe→STN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_STN; i++ ){
            for ( long j = 0; j < per_chanel_N_GPe; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeSTN );
            }
        }
    }
    post_synapse += count;
    count = 0;

    // PTN→STN(集中型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_STN * N_chanel;

    hipMallocManaged ( &n_STN -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→GPe(拡散型)
    post_synapse += N_CMPf * N_GPe;

    // MSN_D1→GPe(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPe; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D1; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND1GPe );
            }
        }
    }
    post_synapse += count;
    count = 0;

    // MSN_D2→GPe(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPe; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D2; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND2GPe );
            }
        }
    }
    post_synapse += count;
    count = 0;

    // STN→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNGPe );
        }
    }
    post_synapse += count;
    count = 0;

    // GPe→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPe );
        }
    }
    post_synapse += count;
    count = 0;

    hipMallocManaged ( &n_GPe -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // CMPf→GPi(拡散型)
    post_synapse += N_CMPf * N_GPi;

    // MSN_D1→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D1; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND1GPi );
            }
        }
    }
    post_synapse += count;
    count = 0;

    // MSN_D2→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D2; j++ ){
                count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND2GPi );
            }
        }
    }
    post_synapse += count;
    count = 0;

    // STN→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_STNGPi );
        }
    }
    post_synapse += count;
    count = 0;

    // GPe→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPi );
        }
    }
    post_synapse += count;
    count = 0;

    hipMallocManaged ( &n_GPi -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // MSN_D1→SNc(拡散型？)
    for ( long i = 0; i < N_SNc; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND1SNc );
        }
    }
    post_synapse += count;
    count = 0;

    // MSN_D2→SNc(拡散型？)
    for ( long i = 0; i < N_SNc; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            count += ( long ) ( ( ( double ) rand () / RAND_MAX ) < P_MSND2SNc );
        }
    }
    post_synapse += count;
    count = 0;

    hipMallocManaged ( &n_SNc -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // PTI→PTN(拡散型)
    post_synapse += N_PTI * N_PTN;

    // PSN→PTN(集中型)
    post_synapse += per_chanel_N_PSN * per_chanel_N_PTN * N_chanel;

    // Th→PTN(集中型?)
    post_synapse += per_chanel_N_Th * per_chanel_N_PTN * N_chanel;

    hipMallocManaged ( &n_PTN -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // PTN→PTI(集中型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_PTI * N_chanel;

    hipMallocManaged ( &n_PTI -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    // GPi→Th(集中型)
    post_synapse += per_chanel_N_GPi * per_chanel_N_Th * N_chanel;

    // PTN→Th(拡散型)
    post_synapse += per_chanel_N_PTN * per_chanel_N_Th * N_chanel;

    hipMallocManaged ( &n_Th -> post, sizeof ( long ) * post_synapse );
    post_synapse = 0;

    srand ( 1 );

    // CMPf→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D1→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND1MSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseMSN_D1;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D2→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND2MSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseMSN_D2;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // FSI→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_FSIMSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseFSI;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // STN→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNMSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→MSN_D1(拡散型)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeMSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // SNc→MSN_D1(拡散型?)
    for ( long i = 0; i < N_MSN_D1; i++ ){
        for ( long j = 0; j < N_SNc; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_SNcMSND1 ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + SynapseSNc;
                count++;
                count_pre++;
            }
        }
        n_MSN_D1 -> num_pre[ i + ( N_MSN_D1 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // PTN→MSN_D1(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN_D1; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_MSN_D1 -> num_pre[ i + ( t * per_chanel_N_MSN_D1 ) + ( N_MSN_D1 * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // PSN→MSN_D1(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN_D1; i++ ){
            for ( long j = 0; j < per_chanel_N_PSN; j++ ){
                n_MSN_D1 -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PSN ) + SynapsePSN;
                count++;
                count_pre++;
            }
            n_MSN_D1 -> num_pre[ i + ( t * per_chanel_N_MSN_D1 ) + ( N_MSN_D1 * k ) ] = count_pre;
        }
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D1→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND1MSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseMSN_D1;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D2→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND2MSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseMSN_D2;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // FSI→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_FSIMSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseFSI;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // STN→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNMSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→MSN_D2(拡散型)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeMSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // SNc→MSN_D2(拡散型?)
    for ( long i = 0; i < N_MSN_D2; i++ ){
        for ( long j = 0; j < N_SNc; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_SNcMSND2 ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + SynapseSNc;
                count++;
                count_pre++;
            }
        }
        n_MSN_D2 -> num_pre[ i + ( N_MSN_D2 * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // PTN→MSN_D2(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN_D2; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_MSN_D2 -> num_pre[ i + ( t * per_chanel_N_MSN_D2 ) + ( N_MSN_D2 * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // PSN→MSN_D2(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_MSN_D2; i++ ){
            for ( long j = 0; j < per_chanel_N_PSN; j++ ){
                n_MSN_D2 -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PSN ) + SynapsePSN;
                count++;
                count_pre++;
            }
            n_MSN_D2 -> num_pre[ i + ( t * per_chanel_N_MSN_D2 ) + ( N_MSN_D2 * k ) ] = count_pre;
        }
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_FSI -> post[ count + post_synapse ] = j  + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // FSI→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_FSI; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_FSIFSI ){
                n_FSI -> post[ count + post_synapse ] = j + SynapseFSI;
                count++;
                count_pre++;
            }
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // STN→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNFSI ){
                n_FSI -> post[ count + post_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→FSI(拡散型)
    for ( long i = 0; i < N_FSI; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeFSI ){
                n_FSI -> post[ count + post_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_FSI -> num_pre[ i + ( N_FSI * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // PTN→FSI(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_FSI; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_FSI -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( t * per_chanel_N_FSI ) + ( N_FSI * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // PSN→FSI(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_FSI; i++ ){
            for ( long j = 0; j < per_chanel_N_PSN; j++ ){
                n_FSI -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PSN ) + SynapsePSN;
                count++;
                count_pre++;
            }
            n_FSI -> num_pre[ i + ( t * per_chanel_N_FSI ) + ( N_FSI * k ) ] = count_pre;
        }
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→STN(拡散型)
    for ( long i = 0; i < N_STN; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_STN -> post[ count + post_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_STN -> num_pre [ i + ( N_STN * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→STN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_STN; i++ ){
            for ( long j = 0; j < per_chanel_N_GPe; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_GPeSTN ){
                    n_STN -> post[ count + post_synapse ] = j + ( t * per_chanel_N_GPe ) + SynapseGPe;
                    count++;
                    count_pre++;
                }
            }
            n_STN -> num_pre[ i + ( t * per_chanel_N_STN ) + ( N_STN * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // PTN→STN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_STN; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_STN -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_STN -> num_pre[ i + ( t * per_chanel_N_STN ) + ( N_STN * k ) ] = count_pre;
        }
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_GPe -> post[ count + post_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D1→GPe(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPe; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D1; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_MSND1GPe ){
                    n_GPe -> post[ count + post_synapse ] = j + ( t * per_chanel_N_MSN_D1 ) + SynapseMSN_D1;
                    count++;
                    count_pre++;
                }
            }
            n_GPe -> num_pre[ i + ( t * per_chanel_N_GPe ) + ( N_GPe * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D2→GPe(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPe; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D2; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_MSND2GPe ){
                    n_GPe -> post[ count + post_synapse ] = j + ( t * per_chanel_N_MSN_D2 ) + SynapseMSN_D2;
                    count++;
                    count_pre++;
                }
            }
            n_GPe -> num_pre[ i + ( t * per_chanel_N_GPe ) + ( N_GPe * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // STN→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNGPe ){
                n_GPe -> post[ count + post_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→GPe(拡散型)
    for ( long i = 0; i < N_GPe; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPe ){
                n_GPe -> post[ count + post_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_GPe -> num_pre[ i + ( N_GPe * k ) ] = count_pre;
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // CMPf→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_CMPf; j++ ){
            n_GPi -> post[ count + post_synapse ] = j + SynapseCMPf;
            count++;
            count_pre++;
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D1→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D1; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_MSND1GPi ){
                    n_GPi -> post[ count + post_synapse ] = j + ( t * per_chanel_N_MSN_D1 ) + SynapseMSN_D1;
                    count++;
                    count_pre++;
                }
            }
            n_GPi -> num_pre[ i + ( t * per_chanel_N_GPi ) + ( N_GPi * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D2→GPi(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_GPi; i++ ){
            for ( long j = 0; j < per_chanel_N_MSN_D2; j++ ){
                if ( ( ( double ) rand () / RAND_MAX ) < P_MSND2GPi ){
                    n_GPi -> post[ count + post_synapse ] = j + ( t * per_chanel_N_MSN_D2 ) + SynapseMSN_D2;
                    count++;
                    count_pre++;
                }
            }
            n_GPi -> num_pre[ i + ( t * per_chanel_N_GPi ) + ( N_GPi * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // STN→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_STN; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_STNGPi ){
                n_GPi -> post[ count + post_synapse ] = j + SynapseSTN;
                count++;
                count_pre++;
            }
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // GPe→GPi(拡散型)
    for ( long i = 0; i < N_GPi; i++ ){
        for ( long j = 0; j < N_GPe; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_GPeGPi ){
                n_GPi -> post[ count + post_synapse ] = j + SynapseGPe;
                count++;
                count_pre++;
            }
        }
        n_GPi -> num_pre[ i + ( N_GPi * k ) ] = count_pre;
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;

    // MSN_D1→SNc(拡散型?)
    for ( long i = 0; i < N_SNc; i++ ){
        for ( long j = 0; j < N_MSN_D1; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND1SNc ){
                n_SNc -> post[ count + post_synapse ] = j + SynapseMSN_D1;
                count++;
                count_pre++;
            }
        }
        n_SNc -> num_pre[ i + ( N_SNc * k ) + 1 ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // MSN_D2→SNc(拡散型?)
    for ( long i = 0; i < N_SNc; i++ ){
        for ( long j = 0; j < N_MSN_D2; j++ ){
            if ( ( ( double ) rand () / RAND_MAX ) < P_MSND2SNc ){
                n_SNc -> post[ count + post_synapse ] = j + SynapseMSN_D2;
                count++;
                count_pre++;
            }
        }
        n_SNc -> num_pre[ i + ( N_SNc * k ) + 1 ] = count_pre;
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;
    n_SNc -> num_pre[ 0 ] = 0;

    // PTI→PTN(拡散型)
    for ( long i = 0; i < N_PTN; i++ ){
        for ( long j = 0; j < N_PTI; j++ ){
            n_PTN -> post[ count + post_synapse ] = j + SynapsePTI;
            count++;
            count_pre++;
        }
        n_PTN -> num_pre[ i + ( N_PTN * k ) + 1 ] = count_pre;
    }
    post_synapse += count;
    count = 0;
    k++;

    // PSN→PTN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_PTN; i++ ){
            for ( long j = 0; j < per_chanel_N_PSN; j++ ){
                n_PTN -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PSN ) + SynapsePSN;
                count++;
                count_pre++;
            }
            n_PTN -> num_pre[ i + ( t * per_chanel_N_PTN ) + ( N_PTN * k ) + 1 ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // Th→PTN(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_PTN; i++ ){
            for ( long j = 0; j < per_chanel_N_Th; j++ ){
                n_PTN -> post[ count + post_synapse ] = j + ( t * per_chanel_N_Th ) + SynapseTh;
                count++;
                count_pre++;
            }
            n_PTN -> num_pre[ i + ( t * per_chanel_N_PTN ) + ( N_PTN * k ) + 1 ] = count_pre;
        }
    }
    post_synapse = 0;
    count = 0;
    count_pre = 0;
    k = 0;
    n_PTN -> num_pre[ 0 ] = 0;

    // PTN→PTI(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_PTI; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_PTI -> post[ count ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_PTI -> num_pre[ i + ( t * per_chanel_N_PTI ) + 1 ] = count_pre;
        }
    }
    count = 0;
    count_pre = 0;
    n_PTI -> num_pre[ 0 ] = 0;

    // GPi→Th(集中型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_Th; i++ ){
            for ( long j = 0; j < per_chanel_N_GPi; j++ ){
                n_Th -> post[ count + post_synapse ] = j + ( t * per_chanel_N_GPi ) + SynapseGPi;
                count++;
                count_pre++;
            }
            n_Th -> num_pre[ i + ( t * per_chanel_N_Th ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    k++;

    // PTN→Th(拡散型)
    for ( int t = 0; t < N_chanel; t++ ){
        for ( long i = 0; i < per_chanel_N_Th; i++ ){
            for ( long j = 0; j < per_chanel_N_PTN; j++ ){
                n_Th -> post[ count + post_synapse ] = j + ( t * per_chanel_N_PTN ) + SynapsePTN;
                count++;
                count_pre++;
            }
            n_Th -> num_pre[ i + ( t * per_chanel_N_Th ) + ( N_Th * k ) ] = count_pre;
        }
    }
    post_synapse += count;
    count = 0;
    count_pre = 0;
    k = 0;
    n_Th -> num_pre[ 0 ] = 0;
}

// メモリ確保や初期化を行う関数を実行する関数
void initalize( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    // 疑似乱数ジェネレータ(pseudo random number generator：PRNG)のシード値の設定
    sfmt_init_gen_rand ( &( n_MSN_D1 -> rng ), 23 );
    sfmt_init_gen_rand ( &( n_MSN_D2 -> rng ), 20 );
    sfmt_init_gen_rand ( &( n_FSI -> rng ), 22 );
    sfmt_init_gen_rand ( &( n_STN -> rng ), 21 );
    sfmt_init_gen_rand ( &( n_GPe -> rng ), 29 );
    sfmt_init_gen_rand ( &( n_GPi -> rng ), 19 );
    sfmt_init_gen_rand ( &( n_SNc -> rng ), 28 );
    sfmt_init_gen_rand ( &( n_PTN -> rng ), 10 );
    sfmt_init_gen_rand ( &( n_PTI -> rng ), 27 );
    sfmt_init_gen_rand ( &( n_PSN -> rng ), 15 );
    sfmt_init_gen_rand ( &( n_Th -> rng ), 17 );
    sfmt_init_gen_rand ( &( n_CMPf -> rng ), 18 );

    // ニューロンを表現する構造体のメンバ変数のメモリ確保
    Allocating_Neuron( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

    // ニューロンを表現する構造体のメンバ変数の初期化
    initalize_Neuron( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );

    // シナプス結合に関するメンバ変数のメモリ確保と乱数を用いて結合の再現
    initalize_synapse( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
}