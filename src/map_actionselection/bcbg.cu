#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include "param.h"
#include "structure.cu"
#include "initalize.cu"
#include "synapse_function.cu"
#include "update.cu"
#include "output.cu"
#include "function.cu"

int main( void ){

  /*
  各ニューロン(以下各ニューロンついて)を再現するためのポインタ変数の定義
  msn_d1→線条体内のD1受容体を発現する中型の有棘細胞(medium spiny neurons：MSN)
  msn_d2→線条体内のD2受容体を発現する中型の有棘細胞(medium spiny neurons：MSN)
  fsi→線条体内の高速でスパイクする介在ニューロン(fast-spiking interneurons：FSI)
  stn→視床下核(subthalamic nucleus：STN)
  gpe→淡蒼球外節(external globus pallidus：GPe)
  gpi→淡蒼球内節(internal globus pallidus：GPi)
  snc→黒質緻密部(substantia nigra pars compacta：SNc)
  ptn→大脳皮質の運動野(pyramidal tract neurons：PTN)
  pti→大脳皮質の運動野の介在ニューロン(pyramidal tract interneurons：PTI)
  psn→大脳皮質の運動野(pyramidal sensory neurons：PSN)
  th→視床(thalamus：Th)
  cmpf→視床下部(centromedian and parafascicular thalamic nuclei：CM/Pf)
  */
  neuron_t *msn_d1, *msn_d2, *fsi, *stn, *gpe, *gpi, *snc, *ptn, *pti, *psn, *th, *cmpf;

  // 定義したポインタ変数のメモリ確保
  hipMallocManaged ( &msn_d1, sizeof ( neuron_t ) );
  hipMallocManaged ( &msn_d2, sizeof ( neuron_t ) );
  hipMallocManaged ( &fsi, sizeof ( neuron_t ) );
  hipMallocManaged ( &stn, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpe, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpi, sizeof ( neuron_t ) );
  hipMallocManaged ( &snc, sizeof ( neuron_t ) );
  hipMallocManaged ( &ptn, sizeof ( neuron_t ) );
  hipMallocManaged ( &pti, sizeof ( neuron_t ) );
  hipMallocManaged ( &psn, sizeof ( neuron_t ) );
  hipMallocManaged ( &th, sizeof ( neuron_t ) );
  hipMallocManaged ( &cmpf, sizeof ( neuron_t ) );


  // シミュレーション用いる変数のメモリ確保・初期化
  initalize ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psn, th, cmpf );

  // 各ニューロンについてのシミュレーション結果を出力するファイルをオープン
  file_open( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psn, th, cmpf  );

  // chanel1(選択肢1)の選択性の変化
  for ( int chanel1 = 0; chanel1 < N_change; chanel1++ ){
    // chanel2(選択肢2)の選択性の変化
    for ( int chanel2 = 0; chanel2 < N_change; chanel2++ ){
      // 1s(1000ms)のシミュレーションを行う関数
      loop ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psn, th, cmpf, chanel1 , chanel2 );
    }
  }

  // メモリ確保した構造体のメンバ変数のメモリの解放
  finalize ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psn, th, cmpf );

  // 出力に使ったファイルをクローズ
  fileclose_firingrate ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psn, th, cmpf );

  // 定義したポインタ変数のメモリ解放
  hipFree ( msn_d1 );
  hipFree ( msn_d2 );
  hipFree ( fsi );
  hipFree ( stn );
  hipFree ( gpe );
  hipFree ( gpi );
  hipFree ( snc );
  hipFree ( ptn );
  hipFree ( pti );
  hipFree ( psn );
  hipFree ( th );
  hipFree ( cmpf );

  return 0;
}