#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include "param.h"
#include "structure.cu"
#include "synapse_function.cu"
#include "update.cu"
#include "output.cu"
#include "function.cu"
#include "initalize.cu"

int main( void ){

  neuron_t *msn_d1, *msn_d2, *fsi, *stn, *gpe, *gpi, *snc, *ptn, *pti, *psa, *th, *cmpf;

  //初期化・メモリの確保・ファイルを開く
  hipMallocManaged ( &msn_d1, sizeof ( neuron_t ) );
  hipMallocManaged ( &msn_d2, sizeof ( neuron_t ) );
  hipMallocManaged ( &fsi, sizeof ( neuron_t ) );
  hipMallocManaged ( &stn, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpe, sizeof ( neuron_t ) );
  hipMallocManaged ( &gpi, sizeof ( neuron_t ) );
  hipMallocManaged ( &snc, sizeof ( neuron_t ) );
  hipMallocManaged ( &ptn, sizeof ( neuron_t ) );
  hipMallocManaged ( &pti, sizeof ( neuron_t ) );
  hipMallocManaged ( &psa, sizeof ( neuron_t ) );
  hipMallocManaged ( &th, sizeof ( neuron_t ) );
  hipMallocManaged ( &cmpf, sizeof ( neuron_t ) );
  
  initalizeNeuron ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psa, th, cmpf );
  initsynapse ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, th );

  //単位時間によるループ(1ms)
  loop ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psa, th, cmpf );

  //メモリの解放・ファイルを閉じる
  finalize ( msn_d1, msn_d2, fsi, stn, gpe, gpi, snc, ptn, pti, psa, th, cmpf );

  hipFree ( msn_d1 );
  hipFree ( msn_d2 );
  hipFree ( fsi );
  hipFree ( stn );
  hipFree ( gpe );
  hipFree ( gpi );
  hipFree ( snc );
  hipFree ( ptn );
  hipFree ( pti );
  hipFree ( psa );
  hipFree ( th );
  hipFree ( cmpf );

  return 0;
}