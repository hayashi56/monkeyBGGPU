#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>

//膜電位の更新
// MSN_D1
__global__ void updatePotential_MSN_D1 ( int nt, neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){

        n_MSN_D1 -> v[ i ] += DT * ( - ( n_MSN_D1 -> v[ i ] ) + Vc_MSN_D1 + ( rho * n_MSN_D1 -> ig[ i ] ) ) / TAU_MSN_D1;
        n_MSN_D1 -> v [ i ] = ( ( n_MSN_D1 -> refr [ i ] ) > 0 ) ? ( V_RESET ) : ( ( n_MSN_D1 -> v [ i ] ) );
        n_MSN_D1 -> s[ i ] = ( n_MSN_D1 -> v[ i ] > THETA_MSN_D1 );
        n_MSN_D1 -> ts[ i ] = ( n_MSN_D1 -> s[ i ] ) * ( nt + 1 ) + ( !( n_MSN_D1 -> s[ i ] ) ) * ( n_MSN_D1 -> ts[ i ] );
        n_MSN_D1 -> v[ i ] = ( n_MSN_D1 -> s[ i ] ) * V_RESET + ( !( n_MSN_D1 -> s[ i ] ) ) * n_MSN_D1 -> v[ i ];
        n_MSN_D1 -> ig[ i ] = V_RESET;
    }
}
// MSN_D2
__global__ void updatePotential_MSN_D2 ( int nt, neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){

        n_MSN_D2 -> v[ i ] += DT * ( - ( n_MSN_D2 -> v[ i ] ) + Vc_MSN_D2 + ( rho * n_MSN_D2 -> ig[ i ] ) ) / TAU_MSN_D2;
        n_MSN_D2 -> s[ i ] = ( n_MSN_D2 -> v[ i ] > THETA_MSN_D2 );
        n_MSN_D2 -> ts[ i ] = ( n_MSN_D2 -> s[ i ] ) * ( nt + 1 ) + ( !( n_MSN_D2 -> s[ i ] ) ) * ( n_MSN_D2 -> ts[ i ] );
        n_MSN_D2 -> v[ i ] = ( n_MSN_D2 -> s[ i ] ) * V_RESET + ( !( n_MSN_D2 -> s[ i ] ) ) * n_MSN_D2 -> v[ i ];
        n_MSN_D2 -> ig[ i ] = V_RESET;
    }
}
// FSI
__global__ void updatePotential_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> v[ i ] += DT * ( - ( n_FSI -> v[ i ] ) + Vc_FSI + ( rho * n_FSI -> ig[ i ] ) ) / TAU_FSI;
        n_FSI -> s[ i ] = ( n_FSI -> v[ i ] > THETA_FSI );
        n_FSI -> ts[ i ] = ( n_FSI -> s[ i ] ) * ( nt + 1 ) + ( !( n_FSI -> s[ i ] ) ) * ( n_FSI -> ts[ i ] );
        n_FSI -> v[ i ] = ( n_FSI -> s[ i ] ) * V_RESET + ( !( n_FSI -> s[ i ] ) ) * n_FSI -> v[ i ];
        n_FSI -> ig[ i ] = V_RESET;
    }
}
// STN
__global__ void updatePotential_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> v[ i ] += DT * ( - ( n_STN -> v[ i ] ) + Vc_STN + ( rho * n_STN -> ig[ i ] ) ) / TAU_STN;
        n_STN -> s[ i ] = ( n_STN -> v[ i ] > THETA_STN );
        n_STN -> ts[ i ] = ( n_STN -> s[ i ]) * ( nt + 1 ) + ( !( n_STN -> s[ i ] ) ) * ( n_STN -> ts[ i ] );
        n_STN -> v[ i ] = ( n_STN -> s[ i ] ) * V_RESET + ( !( n_STN -> s[ i ] ) ) * n_STN -> v[ i ];
        n_STN -> ig[ i ] = V_RESET;
    }
}
// GPe
__global__ void updatePotential_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> v[ i ] += DT * ( - ( n_GPe -> v[ i ] ) + Vc_GPe + ( rho * n_GPe -> ig[ i ] ) ) / TAU_GPe;
        n_GPe -> s[ i ] = ( n_GPe -> v[ i ] > THETA_GPe );
        n_GPe -> ts[ i ] = ( n_GPe -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPe -> s[ i ] ) ) * ( n_GPe -> ts[ i ] );
        n_GPe -> v[ i ] = ( n_GPe -> s[ i ] ) * V_RESET + ( !( n_GPe -> s[ i ] ) ) * n_GPe -> v[ i ];
        n_GPe -> ig[ i ] = V_RESET;
    }
}
// GPi
__global__ void updatePotential_GPi ( int nt, neuron_t *n_GPi ){
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> v[ i ] += DT * ( - ( n_GPi -> v[ i ] ) + Vc_GPi + ( rho * n_GPi -> ig[ i ] ) ) / TAU_GPi;
        n_GPi -> s[ i ] = ( n_GPi -> v[ i ] > THETA_GPi );
        n_GPi -> ts[ i ] = ( n_GPi -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPi -> s[ i ] ) ) * ( n_GPi -> ts[ i ] );
        n_GPi -> v[ i ] = ( n_GPi -> s[ i ] ) * V_RESET + ( !( n_GPi -> s[ i ] ) ) * n_GPi -> v[ i ];
        n_GPi -> ig[ i ] = V_RESET;
    }
}
// SNc
__global__ void updatePotential_SNc ( int nt, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){

        n_SNc -> v[ i ] += DT * ( - ( n_SNc -> v[ i ] ) + Vc_SNc + ( rho * n_SNc -> ig[ i ] ) ) / TAU_SNc;
        n_SNc -> s[ i ] = ( n_SNc -> v[ i ] > THETA_SNc );
        n_SNc -> ts[ i ] = ( n_SNc -> s[ i ] ) * ( nt + 1 ) + ( !( n_SNc -> s[ i ] ) ) * ( n_SNc -> ts[ i ] );
        n_SNc -> v[ i ] = ( n_SNc -> s[ i ] ) * V_RESET + ( !( n_SNc -> s[ i ] ) ) * n_SNc -> v[ i ];
        n_SNc -> ig[ i ] = V_RESET;
    }
}
// PTN
__global__ void updatePotential_PTN ( int nt, neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTN ){

        n_PTN -> v[ i ] += DT * ( - ( n_PTN -> v[ i ] ) + Vc_PTN /*- 91.7*/ + ( rho * n_PTN -> ig[ i ] ) ) / TAU_PTN;
        n_PTN -> s[ i ] = ( n_PTN -> v[ i ] > THETA_PTN );
        n_PTN -> ts[ i ] = ( n_PTN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTN -> s[ i ] ) ) * ( n_PTN -> ts[ i ] );
        n_PTN -> v[ i ] = ( n_PTN -> s[ i ] ) * V_RESET + ( !( n_PTN -> s[ i ] ) ) * n_PTN -> v[ i ];
        n_PTN -> ig[ i ] = V_RESET;
    }
}
// PTI
__global__ void updatePotential_PTI ( int nt, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){

        n_PTI -> v[ i ] += DT * ( - ( n_PTI -> v[ i ] ) + Vc_PTI + ( rho * n_PTI -> ig[ i ] ) ) / TAU_PTI;
        n_PTI -> s[ i ] = ( n_PTI -> v[ i ] > THETA_PTI );
        n_PTI -> ts[ i ] = ( n_PTI -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTI -> s[ i ] ) ) * ( n_PTI -> ts[ i ] );
        n_PTI -> v[ i ] = ( n_PTI -> s[ i ] ) * V_RESET + ( !( n_PTI -> s[ i ] ) ) * n_PTI -> v[ i ];
        n_PTI -> ig[ i ] = V_RESET;
    }
}
// Th
__global__ void updatePotential_Th ( int nt, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_Th ){

        n_Th -> v[ i ] += DT * ( - ( n_Th -> v[ i ] ) + Vc_Th + ( rho * n_Th -> ig[ i ] ) ) / TAU_Th;
        n_Th -> s[ i ] = ( n_Th -> v[ i ] > THETA_Th );
        n_Th -> ts[ i ] = ( n_Th -> s[ i ] ) * ( nt + 1 ) + ( !( n_Th -> s[ i ] ) ) * ( n_Th -> ts[ i ] );
        n_Th -> v[ i ] = ( n_Th -> s[ i ] ) * V_RESET + ( !( n_Th -> s[ i ] ) ) * n_Th -> v[ i ];
        n_Th -> ig[ i ] = V_RESET;
    }
}

void updatePotential ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_Th ){

    updatePotential_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( nt, n_MSN_D1 );
    hipDeviceSynchronize ( );
    updatePotential_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( nt, n_MSN_D2 );
    hipDeviceSynchronize ( );
    updatePotential_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    updatePotential_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    updatePotential_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    updatePotential_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_GPi );
    hipDeviceSynchronize ( );
    updatePotential_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( nt, n_SNc );
    hipDeviceSynchronize ( );
    updatePotential_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN );
    hipDeviceSynchronize ( );
    updatePotential_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( nt, n_PTI );
    hipDeviceSynchronize ( );
    updatePotential_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( nt, n_Th );
    hipDeviceSynchronize ( );
}
//ポアソンモデル
void change_inputneuron ( int nt, neuron_t *n_PSA, neuron_t *n_CMPf, int chanel1, int chanel2, int chanel3, int chanel4, int chanel5, int chanel6, int chanel7, int chanel8, int chanel9, int chanel10 ){


    int i;

    //PSA
    for ( i = 0; i < per_chanel_N_PSA; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel1 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 2; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel2 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 3; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel3 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 4; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel4 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 5; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel5 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 6; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel6 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 7; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel7 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 8; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel8 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < per_chanel_N_PSA * 9; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel9 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }
    for ( ; i < N_PSA; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s [ i ] =  r < ( PHI_MIN_PSA + ( ( ( PHI_MAX_PSA - PHI_MIN_PSA ) * chanel10 ) / N_i ) * n_PSA -> select [ i ] );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }

    //CMPf
    for ( int i = 0; i < N_CMPf; i++ ){

        double r = sfmt_genrand_real2 ( &( n_CMPf -> rng ) );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}

void inputneuron ( int nt, neuron_t *n_PSA, neuron_t *n_CMPf ){

    //PSA
    for ( int i = 0; i < N_PSA; i++ ){

        double r = sfmt_genrand_real2 ( &( n_PSA -> rng ) );
        n_PSA -> s[ i ] = ( r < PHI_MIN_PSA );
        n_PSA -> ts[ i ] = ( n_PSA -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSA -> s[ i ] ) ) * ( n_PSA -> ts[ i ] );
    }

    //CMPf
    for ( int i = 0; i < N_CMPf; i++ ){

        double r = sfmt_genrand_real2 ( &( n_CMPf -> rng ) );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}