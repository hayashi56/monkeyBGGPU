#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include <time.h>

extern "C" { void timer_start( void ); }
extern "C" { double timer_elapsed( void ); }

//不応期
// MSN_D1
__global__ void t_refr_MSN_D1 ( neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){

        n_MSN_D1 -> refr [ i ] = ( n_MSN_D1 -> s [ i ] ) * ( T_REFR ) + ( !( n_MSN_D1 -> s [ i ] ) ) * ( ( n_MSN_D1 -> refr [ i ] ) - 1 ); // set counter
    }
}
// MSN_D2
__global__ void t_refr_MSN_D2 ( neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){

        n_MSN_D2 -> refr [ i ] = ( n_MSN_D2 -> s [ i ] ) * ( T_REFR ) + ( !( n_MSN_D2 -> s [ i ] ) ) * ( ( n_MSN_D2 -> refr [ i ] ) - 1 ); // set counter
    }
}
// FSI
__global__ void t_refr_FSI ( neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> refr [ i ] = ( n_FSI -> s [ i ] ) * ( T_REFR ) + ( !( n_FSI -> s [ i ] ) ) * ( ( n_FSI -> refr [ i ] ) - 1 ); // set counter
    }
}
// STN
__global__ void t_refr_STN ( neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> refr [ i ] = ( n_STN -> s [ i ] ) * ( T_REFR ) + ( !( n_STN -> s [ i ] ) ) * ( ( n_STN -> refr [ i ] ) - 1 ); // set counter
    }
}
// GPe
__global__ void t_refr_GPe ( neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> refr [ i ] = ( n_GPe -> s [ i ] ) * ( T_REFR ) + ( !( n_GPe -> s [ i ] ) ) * ( ( n_GPe -> refr [ i ] ) - 1 ); // set counter
    }
}
// GPi
__global__ void t_refr_GPi ( neuron_t *n_GPi ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> refr [ i ] = ( n_GPi -> s [ i ] ) * ( T_REFR ) + ( !( n_GPi -> s [ i ] ) ) * ( ( n_GPi -> refr [ i ] ) - 1 ); // set counter
    }
}
// SNc
__global__ void t_refr_SNc ( neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){

        n_SNc -> refr [ i ] = ( n_SNc -> s [ i ] ) * ( T_REFR ) + ( !( n_SNc -> s [ i ] ) ) * ( ( n_SNc -> refr [ i ] ) - 1 ); // set counter
    }
}
// PTN
__global__ void t_refr_PTN ( neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTN ){

        n_PTN -> refr [ i ] = ( n_PTN -> s [ i ] ) * ( T_REFR ) + ( !( n_PTN -> s [ i ] ) ) * ( ( n_PTN -> refr [ i ] ) - 1 ); // set counter
    }
}
// PTI
__global__ void t_refr_PTI ( neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){

        n_PTI -> refr [ i ] = ( n_PTI -> s [ i ] ) * ( T_REFR ) + ( !( n_PTI -> s [ i ] ) ) * ( ( n_PTI -> refr [ i ] ) - 1 ); // set counter
    }
}
// Th
__global__ void t_refr_Th ( neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_Th ){

        n_Th -> refr [ i ] = ( n_Th -> s [ i ] ) * ( T_REFR ) + ( !( n_Th -> s [ i ] ) ) * ( ( n_Th -> refr [ i ] ) - 1 ); // set counter
    }
}
void t_refr ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_Th ){

    t_refr_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( n_MSN_D1 );
    hipDeviceSynchronize ( );
    t_refr_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( n_MSN_D2 );
    hipDeviceSynchronize ( );
    t_refr_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( n_FSI );
    hipDeviceSynchronize ( );
    t_refr_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( n_STN );
    hipDeviceSynchronize ( );
    t_refr_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( n_GPe );
    hipDeviceSynchronize ( );
    t_refr_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( n_GPi );
    hipDeviceSynchronize ( );
    t_refr_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( n_SNc );
    hipDeviceSynchronize ( );
    t_refr_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( n_PTN );
    hipDeviceSynchronize ( );
    t_refr_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( n_PTI );
    hipDeviceSynchronize ( );
    t_refr_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( n_Th );
    hipDeviceSynchronize ( );
}

//ループ
void loop( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){
    
    timer_start();
    int nt;

    //FreeRun
    for ( nt = 0; nt < FreeRun; nt++ ){
        updateSynapse ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        InputSynapsePotential ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        updatePotential( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
        input_neuron ( nt, n_PSN, n_CMPf );
        t_refr ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
    }

    for ( ; nt < NT_action1; nt++ ){

        updateSynapse ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        InputSynapsePotential ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        fprintf ( n_MSN_D1 -> file2, "%f %f\n", DT * ( nt + 1 ) - FreeRun, n_MSN_D1 -> ig[ 0 ] );
        fprintf ( n_MSN_D2 -> file2, "%f %f\n", DT * ( nt + 1 ) - FreeRun, n_MSN_D2 -> v[ 0 ] );
        updatePotential( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
        change_input_neuron ( nt, n_PSN, n_CMPf );
        // input_neuron ( nt, n_PSN, n_CMPf );
        t_refr ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_Th );
        output_Spike ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        output_FiringRate_par_seconds ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
        output_Synapse ( nt, n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_SNc, n_PTN, n_PSN, n_CMPf );
    }

    double elapsedTime = timer_elapsed ();

    printf ( "Elapsed time = %f sec.\n", elapsedTime );

    output_FiringRate ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_GPi, n_SNc, n_PTN, n_PTI, n_PSN, n_Th, n_CMPf );
}