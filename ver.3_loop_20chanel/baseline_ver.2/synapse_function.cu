#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "param.h"

__global__ void UpdateConductance_MSN_D1 ( int nt, neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){
        n_MSN_D1 -> conductance_GABA[ i ] = ( float ) PSP_amplitudes_GABA * ( TAU_GABA * n_MSN_D1 -> conductance_GABA[ i ] + ( n_MSN_D1 -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_MSN_D2 ( int nt, neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){
        n_MSN_D2 -> conductance_GABA[ i ] = ( float ) PSP_amplitudes_GABA * ( TAU_GABA * n_MSN_D2 -> conductance_GABA[ i ] + ( n_MSN_D2 -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){
        n_FSI -> conductance_GABA[ i ] = ( float ) PSP_amplitudes_GABA * ( TAU_GABA * n_FSI -> conductance_GABA[ i ] + ( n_FSI -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){
        n_STN -> conductance_AMPA[ i ] = ( float ) PSP_amplitudes_AMPA * ( TAU_AMPA * n_STN -> conductance_AMPA[ i ] + ( n_STN -> ts[ i ] + DELAY == nt + 1 ) );
        n_STN -> conductance_NMDA[ i ] = ( float ) PSP_amplitudes_NMDA * ( TAU_NMDA * n_STN -> conductance_NMDA[ i ] + ( n_STN -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){
        n_GPe -> conductance_GABA[ i ] = ( float ) PSP_amplitudes_GABA * ( TAU_GABA * n_GPe -> conductance_GABA[ i ] + ( n_GPe -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_GPi ( int nt, neuron_t *n_GPi ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){
        n_GPi -> conductance_GABA[ i ] = ( float ) PSP_amplitudes_GABA * ( TAU_GABA * n_GPi -> conductance_GABA[ i ] + ( n_GPi -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_SNc ( int nt, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){
        n_SNc -> conductance_DOPA[ i ] = ( float ) PSP_amplitudes_DOPA * ( TAU_DOPA * n_SNc -> conductance_DOPA[ i ] + ( n_SNc -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_PTN ( int nt, neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_PTN ){
        n_PTN -> conductance_AMPA[ i ] = ( float ) PSP_amplitudes_AMPA * ( TAU_AMPA * n_PTN -> conductance_AMPA[ i ] + ( n_PTN -> ts[ i ] + DELAY == nt + 1 ) );
        n_PTN -> conductance_NMDA[ i ] = ( float ) PSP_amplitudes_NMDA * ( TAU_NMDA * n_PTN -> conductance_NMDA[ i ] + ( n_PTN -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_PTI ( int nt, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){
        n_PTI -> conductance_GABA[ i ] = ( float ) TAU_GABA * n_PTI -> conductance_GABA[ i ] + PSP_amplitudes_GABA * ( n_PTI -> ts[ i ] + DELAY == nt + 1 );
    }
}
__global__ void UpdateConductance_PSN ( int nt, neuron_t *n_PSN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_PSN ){
        n_PSN -> conductance_AMPA[ i ] = ( float ) PSP_amplitudes_AMPA * ( TAU_AMPA * n_PSN -> conductance_AMPA[ i ] + ( n_PSN -> ts[ i ] + DELAY == nt + 1 ) );
        n_PSN -> conductance_NMDA[ i ] = ( float ) PSP_amplitudes_NMDA * ( TAU_NMDA * n_PSN -> conductance_NMDA[ i ] + ( n_PSN -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_Th ( int nt, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N_Th ){
        n_Th -> conductance_AMPA[ i ] = ( float ) PSP_amplitudes_AMPA * ( TAU_AMPA * n_Th -> conductance_AMPA[ i ] + ( n_Th -> ts[ i ] + DELAY == nt + 1 ) );
        n_Th -> conductance_NMDA[ i ] = ( float ) PSP_amplitudes_NMDA * ( TAU_NMDA * n_Th -> conductance_NMDA[ i ] + ( n_Th -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
__global__ void UpdateConductance_CMPf ( int nt, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_CMPf ){
        n_CMPf -> conductance_AMPA[ i ] = ( float ) PSP_amplitudes_AMPA * ( TAU_AMPA * n_CMPf -> conductance_AMPA[ i ] + ( n_CMPf -> ts[ i ] + DELAY == nt + 1 ) );
        n_CMPf -> conductance_NMDA[ i ] = ( float ) PSP_amplitudes_NMDA * ( TAU_NMDA * n_CMPf -> conductance_NMDA[ i ] + ( n_CMPf -> ts[ i ] + DELAY == nt + 1 ) );
    }
}
void UpdateConductance ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    UpdateConductance_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( nt, n_MSN_D1 );
    hipDeviceSynchronize ( );
    UpdateConductance_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( nt, n_MSN_D2 );
    hipDeviceSynchronize ( );
    UpdateConductance_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    UpdateConductance_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    UpdateConductance_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    UpdateConductance_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_GPi );
    hipDeviceSynchronize ( );
    UpdateConductance_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( nt, n_SNc );
    hipDeviceSynchronize ( );
    UpdateConductance_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN );
    hipDeviceSynchronize ( );
    UpdateConductance_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( nt, n_PTI );
    hipDeviceSynchronize ( );
    UpdateConductance_PSN <<< GRID_SIZE_PSN, BLOCK_SIZE >>> ( nt, n_PSN );
    hipDeviceSynchronize ( );
    UpdateConductance_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( nt, n_Th );
    hipDeviceSynchronize ( );
    UpdateConductance_CMPf <<< GRID_SIZE_CMPf, BLOCK_SIZE >>> ( nt, n_CMPf );
    hipDeviceSynchronize ( );
}

// post:MSN_D1
__global__ void Synaptic_current_MSN_D1 ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( n_MSN_D1 -> v[ i ] - rev_potential_AMPA ) * W_CMPfMSND1_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( n_MSN_D1 -> v[ i ] - rev_potential_NMDA ) * W_CMPfMSND1_NMDA ) );
        }
        t++;
        // pre:MSN_D1
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){
            r += - ( n_MSN_D1 -> conductance_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseMSN_D1 ] * ( n_MSN_D1 -> v[ i ] - rev_potential_GABA ) * W_MSND1MSND1 );
        }
        t++;
        // pre:MSN_D2
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){
            r += - ( n_MSN_D2 -> conductance_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseMSN_D2 ] * ( n_MSN_D1 -> v[ i ] - rev_potential_GABA ) * W_MSND2MSND1 );
        }
        t++;
        // pre:FSI
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( n_FSI -> conductance_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseFSI ] * ( n_MSN_D1 -> v[ i ] - rev_potential_GABA ) * W_FSIMSND1 );
        }
        t++;

        // pre:STN
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( ( n_STN -> conductance_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSTN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_AMPA ) * W_STNMSND1_AMPA ) + ( n_STN ->  conductance_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSTN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_NMDA ) * W_STNMSND1_NMDA ) );
        }
        t++;

        // pre:GPe
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseGPe ] * ( n_MSN_D1 -> v[ i ] - rev_potential_GABA ) * W_GPeMSND1 );
        }
        t++;

        // pre:SNc
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( n_SNc -> conductance_DOPA[ n_MSN_D1 -> post[ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapseSNc ] * ( n_MSN_D1 -> v[ i ] - rev_potential_DOPA ) * W_SNcMSND1 );
        }
        t++;

        // pre:PTN
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePTN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_AMPA ) * W_PTNMSND1_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePTN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_NMDA ) * W_PTNMSND1_NMDA ) );
        }
        t++;

        // pre:PSN
        for ( int j = 0; j < ( n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) ] - n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ); j++ ){ 
            r += - ( ( n_PSN -> conductance_AMPA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePSN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_AMPA ) * W_PSNMSND1_AMPA ) + ( n_PSN ->  conductance_NMDA[ n_MSN_D1 -> post [ j + n_MSN_D1 -> num_pre[ i + ( t * N_MSN_D1 ) - 1 ] ] - SynapsePSN ] * ( n_MSN_D1 -> v[ i ] - rev_potential_NMDA ) * W_PSNMSND1_NMDA ) );
        }
        n_MSN_D1 -> i_syn[ i ] = r;
    }
}
// post:MSN_D2
__global__ void Synaptic_current_MSN_D2 ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( n_MSN_D2 -> v[ i ] - rev_potential_AMPA ) * W_CMPfMSND2_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( n_MSN_D2 -> v[ i ] - rev_potential_NMDA ) * W_CMPfMSND2_NMDA ) );
        }
        t++;
        // pre:MSN_D1
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){
            r += - ( n_MSN_D1 -> conductance_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseMSN_D1 ] * ( n_MSN_D2 -> v[ i ] - rev_potential_GABA ) * W_MSND1MSND2 );
        }
        t++;
        // pre:MSN_D2
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){
            r += - ( n_MSN_D2 -> conductance_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseMSN_D2 ] * ( n_MSN_D2 -> v[ i ] - rev_potential_GABA ) * W_MSND2MSND2 );
        }
        t++;
        // pre:FSI
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( n_FSI -> conductance_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseFSI ] * ( n_MSN_D2 -> v[ i ] - rev_potential_GABA ) * W_FSIMSND2 );
        }
        t++;

        // pre:STN
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( ( n_STN -> conductance_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSTN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_AMPA ) * W_STNMSND2_AMPA ) + ( n_STN ->  conductance_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSTN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_NMDA ) * W_STNMSND2_NMDA ) );
        }
        t++;

        // pre:GPe
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseGPe ] * ( n_MSN_D2 -> v[ i ] - rev_potential_GABA ) * W_GPeMSND2 );
        }
        t++;

        // pre:SNc
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( n_SNc -> conductance_DOPA[ n_MSN_D2 -> post[ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapseSNc ] * ( n_MSN_D2 -> v[ i ] - rev_potential_DOPA ) * W_SNcMSND2 );
        }
        t++;

        // pre:PTN
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePTN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_AMPA ) * W_PTNMSND2_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePTN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_NMDA ) * W_PTNMSND2_NMDA ) );
        }
        t++;

        // pre:PSN
        for ( int j = 0; j < ( n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) ] - n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ); j++ ){ 
            r += - ( ( n_PSN -> conductance_AMPA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePSN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_AMPA ) * W_PSNMSND2_AMPA ) + ( n_PSN ->  conductance_NMDA[ n_MSN_D2 -> post [ j + n_MSN_D2 -> num_pre[ i + ( t * N_MSN_D2 ) - 1 ] ] - SynapsePSN ] * ( n_MSN_D2 -> v[ i ] - rev_potential_NMDA ) * W_PSNMSND2_NMDA ) );
        }
        n_MSN_D2 -> i_syn[ i ] = r;
    }
}
// post:FSI
__global__ void Synaptic_current_FSI ( neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_PTN, neuron_t *n_PSN, neuron_t *n_CMPf ){
    
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_FSI ) {
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( n_FSI -> v[ i ] - rev_potential_AMPA ) * W_CMPfFSI_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( n_FSI -> v[ i ] - rev_potential_NMDA ) * W_CMPfFSI_NMDA ) );
        }
        t++;
        // pre:FSI
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += - ( n_FSI -> conductance_GABA[ n_FSI -> post[ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseFSI ] * ( n_FSI -> v[ i ] - rev_potential_GABA ) * W_FSIFSI );
        }
        t++;
        // pre:STN
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += - ( ( n_STN -> conductance_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] * ( n_FSI -> v[ i ] - rev_potential_AMPA ) * W_STNFSI_AMPA ) + ( n_STN ->  conductance_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseSTN ] * ( n_FSI -> v[ i ] - rev_potential_NMDA ) * W_STNFSI_NMDA ) );
        }
        t++;
        // pre:GPe
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_FSI -> post[ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapseGPe ] * ( n_FSI -> v[ i ] - rev_potential_GABA ) * W_GPeFSI );
        }
        t++;
        // pre:PTN
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] * ( n_FSI -> v[ i ] - rev_potential_AMPA ) * W_PTNFSI_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePTN ] * ( n_FSI -> v[ i ] - rev_potential_NMDA ) * W_PTNFSI_NMDA ) );
        }
        t++;
        // pre:PSN
        for ( int j = 0; j < ( n_FSI -> num_pre[ i + ( t * N_FSI ) ] - n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ); j++ ){ 
            r += - ( ( n_PSN -> conductance_AMPA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePSN ] * ( n_FSI -> v[ i ] - rev_potential_AMPA ) * W_PSNFSI_AMPA ) + ( n_PSN ->  conductance_NMDA[ n_FSI -> post [ j + n_FSI -> num_pre[ i + ( t * N_FSI ) - 1 ] ] - SynapsePSN ] * ( n_FSI -> v[ i ] - rev_potential_NMDA ) * W_PSNFSI_NMDA ) );
        }
        n_FSI -> i_syn[ i ] = r;
    }
}
// post:STN
__global__ void Synaptic_current_STN ( neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_PTN, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_STN ){
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( n_STN -> v[ i ] - rev_potential_AMPA ) * W_CMPfSTN_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( n_STN -> v[ i ] - rev_potential_NMDA ) * W_CMPfSTN_NMDA ) );
        }
        t++;
        // pre:GPe
        for ( int j = 0; j < ( n_STN -> num_pre[ i + ( t * N_STN ) ] - n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_STN -> post[ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapseGPe ] * ( n_STN -> v[ i ] - rev_potential_GABA ) * W_GPeSTN );
        }
        t++;
        // pre:PTN
        for ( int j = 0; j < ( n_STN -> num_pre[ i + ( t * N_STN ) ] - n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] * ( n_STN -> v[ i ] - rev_potential_AMPA ) * W_PTNSTN_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_STN -> post [ j + n_STN -> num_pre[ i + ( t * N_STN ) - 1 ] ] - SynapsePTN ] * ( n_STN -> v[ i ] - rev_potential_NMDA ) * W_PTNSTN_NMDA ) );
        }
        t++;
        n_STN -> i_syn[ i ] = r;
    }
}
// post:GPe
__global__ void Synaptic_current_GPe ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_GPe ){
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( - n_GPe -> v[ i ] + rev_potential_AMPA ) * W_CMPfGPe_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( - n_GPe -> v[ i ] + rev_potential_NMDA ) * W_CMPfGPe_NMDA ) );
        }
        t++;
        // pre:MSN_D1
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){
            r += - ( n_MSN_D1 -> conductance_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_MSND1GPe );
        }
        t++;
        // pre:MSN_D2
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){
            r += - ( n_MSN_D2 -> conductance_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_MSND2GPe );
        }
        t++;
        // pre:STN
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r += - ( ( n_STN -> conductance_AMPA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] * ( - n_GPe -> v[ i ] + rev_potential_AMPA ) * W_STNGPe_AMPA ) + ( n_STN ->  conductance_NMDA[ n_GPe -> post [ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseSTN ] * ( - n_GPe -> v[ i ] + rev_potential_NMDA ) * W_STNGPe_NMDA ) );
        }
        t++;
        // pre:GPe
        for ( int j = 0; j < ( n_GPe -> num_pre[ i + ( t * N_GPe ) ] - n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_GPe -> post[ j + n_GPe -> num_pre[ i + ( t * N_GPe ) - 1 ] ] - SynapseGPe ] * ( - n_GPe -> v[ i ] + rev_potential_GABA ) * W_GPeGPe );
        }
        n_GPe -> i_syn[ i ] = r;
    }
}
// post:GPi
__global__ void Synaptic_current_GPi ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_CMPf ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_GPi ){
        float r = 0;
        int t = 0;
        // pre:CMPf
        for ( int j = 0; j < N_CMPf; j++ ){ 
            r += - ( ( n_CMPf -> conductance_AMPA[ j ] * ( - n_GPi -> v[ i ] + rev_potential_AMPA ) * W_CMPfGPi_AMPA ) + ( n_CMPf ->  conductance_NMDA[ j ] * ( - n_GPi -> v[ i ] + rev_potential_NMDA ) * W_CMPfGPi_NMDA ) );
        }
        t++;
        // pre:MSN_D1
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){
            r += - ( n_MSN_D1 -> conductance_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN_D1 ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_MSND1GPi );
        }
        t++;
        // pre:MSN_D2
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){
            r += - ( n_MSN_D2 -> conductance_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseMSN_D2 ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_MSND2GPi );
        }
        t++;
        // pre:STN
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r += - ( ( n_STN -> conductance_AMPA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] * ( - n_GPi -> v[ i ] + rev_potential_AMPA ) * W_STNGPi_AMPA ) + ( n_STN ->  conductance_NMDA[ n_GPi -> post [ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseSTN ] * ( - n_GPi -> v[ i ] + rev_potential_NMDA ) * W_STNGPi_NMDA ) );
        }
        t++;
        // pre:GPe
        for ( int j = 0; j < ( n_GPi -> num_pre[ i + ( t * N_GPi ) ] - n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ); j++ ){ 
            r += - ( n_GPe -> conductance_GABA[ n_GPi -> post[ j + n_GPi -> num_pre[ i + ( t * N_GPi ) - 1 ] ] - SynapseGPe ] * ( - n_GPi -> v[ i ] + rev_potential_GABA ) * W_GPeGPi );
        }
        n_GPi -> i_syn[ i ] = r;
    }
}
// post:SNc
__global__ void Synaptic_current_SNc ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){
        float r = 0;
        int t = 0;
        // pre:MSN_D1
        for ( int j = 0; j < ( n_SNc -> num_pre[ i + 1 ] - n_SNc -> num_pre[ i ] ); j++ ){ 
            r += - ( n_MSN_D1 -> conductance_GABA[ n_SNc -> post[ j + n_SNc -> num_pre[ i ] ] - SynapseMSN_D1 ] * ( - n_SNc -> v[ i ] + rev_potential_GABA ) * W_MSND1SNc );
        }
        t++;
        // pre:MSN_D2
        for ( int j = 0; j < ( n_SNc -> num_pre[ i + ( t * N_SNc ) + 1 ] - n_SNc -> num_pre[ i + ( t * N_SNc ) ] ); j++ ){
            r += - ( n_MSN_D2 -> conductance_GABA[ n_SNc -> post[ j + n_SNc -> num_pre[ i + ( t * N_SNc ) ] ] - SynapseMSN_D2 ] * ( - n_SNc -> v[ i ] + rev_potential_GABA ) * W_MSND2SNc );
        }
        n_SNc -> i_syn[ i ] = r;
    }
}
// post:PTN
__global__ void Synaptic_current_PTN ( neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_PTN ){
        float r = 0;
        int t = 0;
        // pre:PTI
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + 1 ] - n_PTN -> num_pre[ i ] ); j++ ){
            r += - ( n_PTI -> conductance_GABA[ n_PTN -> post[ j + n_PTN -> num_pre[ i ] ] - SynapsePTI ] * ( - n_PTN -> v[ i ] + rev_potential_GABA ) * W_PTIPTN );
        }
        t++;
        // pre:PSN
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + ( t * N_PTN ) + 1 ] - n_PTN -> num_pre[ i + ( t * N_PTN ) ] ); j++ ){ 
            r += - ( ( n_PSN -> conductance_AMPA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapsePSN ] * ( - n_PTN -> v[ i ] + rev_potential_AMPA ) * W_PSNPTN_AMPA ) + ( n_PSN ->  conductance_NMDA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapsePSN ] * ( - n_PTN -> v[ i ] + rev_potential_NMDA ) * W_PSNPTN_NMDA ) );
        }
        t++;
        // pre:Th
        for ( int j = 0; j < ( n_PTN -> num_pre[ i + ( t * N_PTN ) + 1 ] - n_PTN -> num_pre[ i + ( t * N_PTN ) ] ); j++ ){ 
            r += - ( ( n_Th -> conductance_AMPA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapseTh ] * ( - n_PTN -> v[ i ] + rev_potential_AMPA ) * W_ThPTN_AMPA ) + ( n_Th ->  conductance_NMDA[ n_PTN -> post [ j + n_PTN -> num_pre[ i + ( t * N_PTN ) - 1 ] ] - SynapseTh ] * ( - n_PTN -> v[ i ] + rev_potential_NMDA ) * W_ThPTN_NMDA ) );
        }
        n_PTN -> i_syn[ i ] = r;
    }
}
// post:PTI
__global__ void Synaptic_current_PTI ( neuron_t *n_PTN, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    

    if ( i < N_PTI ){
        float r = 0;
        // pre:PTN
        for ( int j = 0; j < ( n_PTI -> num_pre[ i + 1 ] - n_PTI -> num_pre[ i ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_PTI -> post [ j + n_PTI -> num_pre[ i ] ] - SynapsePTN ] * ( - n_PTI -> v[ i ] + rev_potential_AMPA ) * W_PTNPTI_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_PTI -> post [ j + n_PTI -> num_pre[ i ] ] - SynapsePTN ] * ( - n_PTI -> v[ i ] + rev_potential_NMDA ) * W_PTNPTI_NMDA ) );
        }
        n_PTI -> i_syn[ i ] = r;
    }
}
// post:Th
__global__ void Synaptic_current_Th ( neuron_t *n_GPi, neuron_t *n_PTN, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if ( i < N_Th ){
        float r = 0;
        int t = 0;
        // pre:GPi
        for ( int j = 0; j < ( n_Th -> num_pre[ i + 1 ] - n_Th -> num_pre[ i ] ); j++ ){
            r += - ( n_GPi -> conductance_GABA[ n_Th -> post[ j + n_Th -> num_pre[ i ] ] - SynapseGPi ] * ( - n_Th -> v[ i ] + rev_potential_GABA ) * W_GPiTh );
        }
        t++;
        // pre:PTN
        for ( int j = 0; j < ( n_Th -> num_pre[ i + ( t * N_Th ) + 1 ] - n_Th -> num_pre[ i + ( t * N_Th ) ] ); j++ ){ 
            r += - ( ( n_PTN -> conductance_AMPA[ n_Th -> post [ j + n_Th -> num_pre[ i + ( t * N_Th ) - 1 ] ] - SynapsePTN ] * ( - n_Th -> v[ i ] + rev_potential_AMPA ) * W_PTNTh_AMPA ) + ( n_PTN ->  conductance_NMDA[ n_Th -> post [ j + n_Th -> num_pre[ i + ( t * N_Th ) - 1 ] ] - SynapsePTN ] * ( - n_Th -> v[ i ] + rev_potential_NMDA ) * W_PTNTh_NMDA ) );
        }
        n_Th -> i_syn[ i ] = r;
    }
}

void Synaptic_current ( neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_PSN, neuron_t *n_Th, neuron_t *n_CMPf ){

    Synaptic_current_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_SNc, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( n_MSN_D1, n_MSN_D2, n_FSI, n_STN, n_GPe, n_SNc, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( n_FSI, n_STN, n_GPe, n_PTN, n_PSN, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( n_STN, n_GPe, n_PTN, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( n_MSN_D1, n_MSN_D2, n_STN, n_GPe, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( n_MSN_D1, n_MSN_D2, n_STN, n_GPe, n_GPi, n_CMPf );
    hipDeviceSynchronize ( );
    Synaptic_current_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( n_MSN_D1, n_MSN_D2, n_SNc );
    hipDeviceSynchronize ( );
    Synaptic_current_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( n_PTN, n_PTI, n_PSN, n_Th );
    hipDeviceSynchronize ( );
    Synaptic_current_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( n_PTN, n_PTI );
    hipDeviceSynchronize ( );
    Synaptic_current_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( n_GPi, n_PTN, n_Th );
    hipDeviceSynchronize ( );
}