#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <stdbool.h>
#include "param.h"

// 下記は各ニューロンについて積分発火モデル(Leaky integrate-and-fire model)における膜電位の更新や膜電位をもとに発火の有無を判定をGPUで並列計算する関数
__global__ void updatePotential_MSN_D1 ( int nt, neuron_t *n_MSN_D1 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D1 ){

        n_MSN_D1 -> v[ i ] += DT * ( - ( n_MSN_D1 -> v[ i ] - Er_MSN_D1 ) + Vc_MSN_D1 + ( rho * n_MSN_D1 -> i_syn[ i ] ) ) / TAU_MSN_D1;
        n_MSN_D1 -> s[ i ] = ( n_MSN_D1 -> v[ i ] > THETA_MSN_D1 ) * ( ( n_MSN_D1 -> refr[ i ] ) <= 0 );
        n_MSN_D1 -> ts[ i ] = ( n_MSN_D1 -> s[ i ] ) * ( nt + 1 ) + ( !( n_MSN_D1 -> s[ i ] ) ) * ( n_MSN_D1 -> ts[ i ] );
        n_MSN_D1 -> v[ i ] = ( n_MSN_D1 -> s[ i ] ) * Er_MSN_D1 + ( !( n_MSN_D1 -> s[ i ] ) ) * n_MSN_D1 -> v[ i ];
        n_MSN_D1 -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_MSN_D2 ( int nt, neuron_t *n_MSN_D2 ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_MSN_D2 ){

        n_MSN_D2 -> v[ i ] += DT * ( - ( n_MSN_D2 -> v[ i ] - Er_MSN_D2 ) + Vc_MSN_D2 + ( rho * n_MSN_D2 -> i_syn[ i ] ) ) / TAU_MSN_D2;
        n_MSN_D2 -> s[ i ] = ( n_MSN_D2 -> v[ i ] > THETA_MSN_D2 ) * ( ( n_MSN_D2 -> refr[ i ] ) <= 0 );
        n_MSN_D2 -> ts[ i ] = ( n_MSN_D2 -> s[ i ] ) * ( nt + 1 ) + ( !( n_MSN_D2 -> s[ i ] ) ) * ( n_MSN_D2 -> ts[ i ] );
        n_MSN_D2 -> v[ i ] = ( n_MSN_D2 -> s[ i ] ) * Er_MSN_D2 + ( !( n_MSN_D2 -> s[ i ] ) ) * n_MSN_D2 -> v[ i ];
        n_MSN_D2 -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_FSI ( int nt, neuron_t *n_FSI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_FSI ){

        n_FSI -> v[ i ] += DT * ( - ( n_FSI -> v[ i ] - Er_FSI ) + Vc_FSI + ( rho * n_FSI -> i_syn[ i ] ) ) / TAU_FSI;
        n_FSI -> s[ i ] = ( n_FSI -> v[ i ] > THETA_FSI ) * ( ( n_FSI -> refr[ i ] ) <= 0 );
        n_FSI -> ts[ i ] = ( n_FSI -> s[ i ] ) * ( nt + 1 ) + ( !( n_FSI -> s[ i ] ) ) * ( n_FSI -> ts[ i ] );
        n_FSI -> v[ i ] = ( n_FSI -> s[ i ] ) * Er_FSI + ( !( n_FSI -> s[ i ] ) ) * n_FSI -> v[ i ];
        n_FSI -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_STN ( int nt, neuron_t *n_STN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_STN ){

        n_STN -> v[ i ] += DT * ( - ( n_STN -> v[ i ] - Er_STN ) + Vc_STN + ( rho * n_STN -> i_syn[ i ] ) ) / TAU_STN;
        n_STN -> s[ i ] = ( n_STN -> v[ i ] > THETA_STN ) * ( ( n_STN -> refr [ i ] ) <= 0 );
        n_STN -> ts[ i ] = ( n_STN -> s[ i ]) * ( nt + 1 ) + ( !( n_STN -> s[ i ] ) ) * ( n_STN -> ts[ i ] );
        n_STN -> v[ i ] = ( n_STN -> s[ i ] ) * Er_STN + ( !( n_STN -> s[ i ] ) ) * n_STN -> v[ i ];
        n_STN -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_GPe ( int nt, neuron_t *n_GPe ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPe ){

        n_GPe -> v[ i ] += DT * ( - ( n_GPe -> v[ i ] - Er_GPe ) + Vc_GPe + ( rho * n_GPe -> i_syn[ i ] ) ) / TAU_GPe;
        n_GPe -> s[ i ] = ( n_GPe -> v[ i ] > THETA_GPe ) * ( ( n_GPe -> refr [ i ] ) <= 0 );
        n_GPe -> ts[ i ] = ( n_GPe -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPe -> s[ i ] ) ) * ( n_GPe -> ts[ i ] );
        n_GPe -> v[ i ] = ( n_GPe -> s[ i ] ) * Er_GPe + ( !( n_GPe -> s[ i ] ) ) * n_GPe -> v[ i ];
        n_GPe -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_GPi ( int nt, neuron_t *n_GPi ){
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_GPi ){

        n_GPi -> v[ i ] += DT * ( - ( n_GPi -> v[ i ] - Er_GPi ) + Vc_GPi + ( rho * n_GPi -> i_syn[ i ] ) ) / TAU_GPi;
        n_GPi -> s[ i ] = ( n_GPi -> v[ i ] > THETA_GPi ) * ( ( n_GPi -> refr [ i ] ) <= 0 );
        n_GPi -> ts[ i ] = ( n_GPi -> s[ i ] ) * ( nt + 1 ) + ( !( n_GPi -> s[ i ] ) ) * ( n_GPi -> ts[ i ] );
        n_GPi -> v[ i ] = ( n_GPi -> s[ i ] ) * Er_GPi + ( !( n_GPi -> s[ i ] ) ) * n_GPi -> v[ i ];
        n_GPi -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_SNc ( int nt, neuron_t *n_SNc ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_SNc ){

        n_SNc -> v[ i ] += DT * ( - ( n_SNc -> v[ i ] - Er_SNc ) + Vc_SNc + ( rho * n_SNc -> i_syn[ i ] ) ) / TAU_SNc;
        n_SNc -> s[ i ] = ( n_SNc -> v[ i ] > THETA_SNc ) * ( ( n_SNc -> refr [ i ] ) <= 0 );
        n_SNc -> ts[ i ] = ( n_SNc -> s[ i ] ) * ( nt + 1 ) + ( !( n_SNc -> s[ i ] ) ) * ( n_SNc -> ts[ i ] );
        n_SNc -> v[ i ] = ( n_SNc -> s[ i ] ) * Er_SNc + ( !( n_SNc -> s[ i ] ) ) * n_SNc -> v[ i ];
        n_SNc -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_PTN ( int nt, neuron_t *n_PTN ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTN ){

        n_PTN -> v[ i ] += DT * ( - ( n_PTN -> v[ i ] - Er_PTN ) + Vc_PTN + ( rho * n_PTN -> i_syn[ i ] ) ) / TAU_PTN;
        n_PTN -> s[ i ] = ( n_PTN -> v[ i ] > THETA_PTN ) * ( ( n_PTN -> refr [ i ] ) <= 0 );
        n_PTN -> ts[ i ] = ( n_PTN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTN -> s[ i ] ) ) * ( n_PTN -> ts[ i ] );
        n_PTN -> v[ i ] = ( n_PTN -> s[ i ] ) * Er_PTN + ( !( n_PTN -> s[ i ] ) ) * n_PTN -> v[ i ];
        n_PTN -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_PTI ( int nt, neuron_t *n_PTI ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_PTI ){

        n_PTI -> v[ i ] += DT * ( - ( n_PTI -> v[ i ] - Er_PTI ) + Vc_PTI + ( rho * n_PTI -> i_syn[ i ] ) ) / TAU_PTI;
        n_PTI -> s[ i ] = ( n_PTI -> v[ i ] > THETA_PTI && n_PTI -> refr [ i ] <= 0 );
        n_PTI -> ts[ i ] = ( n_PTI -> s[ i ] ) * ( nt + 1 ) + ( !( n_PTI -> s[ i ] ) ) * ( n_PTI -> ts[ i ] );
        n_PTI -> v[ i ] = ( n_PTI -> s[ i ] ) * Er_PTI + ( !( n_PTI -> s[ i ] ) ) * n_PTI -> v[ i ];
        n_PTI -> i_syn[ i ] = PSP_RESET;
    }
}
__global__ void updatePotential_Th ( int nt, neuron_t *n_Th ){

    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if ( i < N_Th ){

        n_Th -> v[ i ] += DT * ( - ( n_Th -> v[ i ] - Er_Th ) + Vc_Th + ( rho * n_Th -> i_syn[ i ] ) ) / TAU_Th;
        n_Th -> s[ i ] = ( n_Th -> v[ i ] > THETA_Th ) * ( ( n_Th -> refr [ i ] ) <= 0 );
        n_Th -> ts[ i ] = ( n_Th -> s[ i ] ) * ( nt + 1 ) + ( !( n_Th -> s[ i ] ) ) * ( n_Th -> ts[ i ] );
        n_Th -> v[ i ] = ( n_Th -> s[ i ] ) * Er_Th + ( !( n_Th -> s[ i ] ) ) * n_Th -> v[ i ];
        n_Th -> i_syn[ i ] = PSP_RESET;
    }
}


// 下記はLIF modelでの更新の関数をCPUで各ニューロンについて実行する関数
void updatePotential ( int nt, neuron_t *n_MSN_D1, neuron_t *n_MSN_D2, neuron_t *n_FSI, neuron_t *n_STN, neuron_t *n_GPe, neuron_t *n_GPi, neuron_t *n_SNc, neuron_t *n_PTN, neuron_t *n_PTI, neuron_t *n_Th ){

    updatePotential_MSN_D1 <<< GRID_SIZE_MSN_D1, BLOCK_SIZE >>> ( nt, n_MSN_D1 );
    hipDeviceSynchronize ( );
    updatePotential_MSN_D2 <<< GRID_SIZE_MSN_D2, BLOCK_SIZE >>> ( nt, n_MSN_D2 );
    hipDeviceSynchronize ( );
    updatePotential_FSI <<< GRID_SIZE_FSI, BLOCK_SIZE >>> ( nt, n_FSI );
    hipDeviceSynchronize ( );
    updatePotential_STN <<< GRID_SIZE_STN, BLOCK_SIZE >>> ( nt, n_STN );
    hipDeviceSynchronize ( );
    updatePotential_GPe <<< GRID_SIZE_GPe, BLOCK_SIZE >>> ( nt, n_GPe );
    hipDeviceSynchronize ( );
    updatePotential_GPi <<< GRID_SIZE_GPi, BLOCK_SIZE >>> ( nt, n_GPi );
    hipDeviceSynchronize ( );
    updatePotential_SNc <<< GRID_SIZE_SNc, BLOCK_SIZE >>> ( nt, n_SNc );
    hipDeviceSynchronize ( );
    updatePotential_PTN <<< GRID_SIZE_PTN, BLOCK_SIZE >>> ( nt, n_PTN );
    hipDeviceSynchronize ( );
    updatePotential_PTI <<< GRID_SIZE_PTI, BLOCK_SIZE >>> ( nt, n_PTI );
    hipDeviceSynchronize ( );
    updatePotential_Th <<< GRID_SIZE_Th, BLOCK_SIZE >>> ( nt, n_Th );
    hipDeviceSynchronize ( );
}


/*
下記は外部入力を担っているニューロンをポアソンスパイクで再現した際の発火の有無を判定する関数で、
change_input_neuronは入力を安静時の発火率から変化させ、入力に選択肢の選択性を提示する関数
input_neuronは入力を安静時の発火率で行う関数
*/
void input ( int nt, neuron_t *n_PSN, neuron_t *n_CMPf, int channel ){

    if ( channel == 9 || channel == 10 ){
        for ( int i = 0 + ( per_chanel_N_PTN * channel ); i < per_chanel_N_PTN  * ( channel + 1 ); i++ ){

            double r = sfmt_genrand_real2 ( &n_PSN -> rng );
            n_PSN -> s[ i ] =  r < ( PHI_MIN_PSN + ( ( ( PHI_MAX_PSN - PHI_MIN_PSN ) * CHANGE_FR ) / N_i ) * n_PSN -> select[ i ] );
            n_PSN -> ts[ i ] = ( n_PSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSN -> s[ i ] ) ) * ( n_PSN -> ts[ i ] );
        }
    }
    else{
        for ( int i = 0 + ( per_chanel_N_PTN * channel ); i < per_chanel_N_PTN  * ( channel + 1 ); i++ ){

            double r = sfmt_genrand_real2 ( &n_PSN -> rng );
            n_PSN -> s[ i ] =  r < ( PHI_MIN_PSN + ( ( ( PHI_MAX_PSN - PHI_MIN_PSN ) * REST_FR ) / N_i ) * n_PSN -> select[ i ] );
            n_PSN -> ts[ i ] = ( n_PSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSN -> s[ i ] ) ) * ( n_PSN -> ts[ i ] );
        }
    }
}

void change_input_neuron ( int nt, neuron_t *n_PSN, neuron_t *n_CMPf ){

    //sensory cortexについて
    for ( int channel = 0; channel < 20; channel++ ){
        
        input( nt, n_PSN, n_CMPf, channel );
    }
    //CMPfについて
    for ( int i = 0; i < N_CMPf; i++ ){

        double r = sfmt_genrand_real2 ( &n_CMPf -> rng );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}

void input_neuron ( int nt, neuron_t *n_PSN, neuron_t *n_CMPf ){

    //PSNについて
    for ( int i = 0; i < N_PSN; i++ ){

        double r = sfmt_genrand_real2 ( &n_PSN -> rng );
        n_PSN -> s[ i ] = ( r < PHI_MIN_PSN );
        n_PSN -> ts[ i ] = ( n_PSN -> s[ i ] ) * ( nt + 1 ) + ( !( n_PSN -> s[ i ] ) ) * ( n_PSN -> ts[ i ] );
    }
    //CMPfについて
    for ( int i = 0; i < N_CMPf; i++ ){

        double r = sfmt_genrand_real2 ( &n_CMPf -> rng );
        n_CMPf -> s[ i ] = ( r < PHI_CMPf );
        n_CMPf -> ts[ i ] = ( n_CMPf -> s[ i ] ) * ( nt + 1 ) + ( !( n_CMPf -> s[ i ] ) ) * ( n_CMPf -> ts[ i ] );
    }
}